#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "math_functions.h"
#include "round.cuh"

#include <cmath>
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include "utils.h"
#include <thrust\device_ptr.h>
#include <thrust\sort.h>
#include <map>
#include <string>
#include <sstream>

//#pragma comment(lib, "hiprand.lib")

using std::map;
using std::string;
using std::stringstream;

#define INDEX_SIZE_IN_BYTES 8
#define EXTRACT_9 0x7fffffffffffffff
#define EXTRACT_8 0x00ffffffffffffff
#define EXTRACT_7 0x0001ffffffffffff
#define EXTRACT_6 0x000003ffffffffff

#define ROTR32(x, n)  (((0U + (x)) << (32 - (n))) | ((x) >> (n)))  // Assumes that x is uint32_t and 0 < n < 32

#define LOADSCHEDULE(i)  \
		schedule[i] = (uint32_t)data[i * 4 + 0] << 24  \
		            | (uint32_t)data[i * 4 + 1] << 16  \
		            | (uint32_t)data[i * 4 + 2] <<  8  \
		            | (uint32_t)data[i * 4 + 3] <<  0;

#define SCHEDULE(i)  \
		schedule[i] = 0U + schedule[i - 16] + schedule[i - 7]  \
			+ (ROTR32(schedule[i - 15], 7) ^ ROTR32(schedule[i - 15], 18) ^ (schedule[i - 15] >> 3))  \
			+ (ROTR32(schedule[i - 2], 17) ^ ROTR32(schedule[i - 2], 19) ^ (schedule[i - 2] >> 10));

//#define SCHEDULE(i) \
//	asm("{\n\t" \
//		".reg .u32 t1;\n\t" \
//		".reg .u32 t2;\n\t" \
//		".reg .u32 t3;\n\t" \
//		".reg .u32 s1;\n\t" \
//		".reg .u32 s2;\n\t" \
//		".reg .u32 s3;\n\t" \
//		".reg .u32 s4;\n\t" \
//		"mov.u32 s1, %1;\n\t" \
//		"mov.u32 s2, %2;\n\t" \
//		"mov.u32 s3, %3;\n\t" \
//		"mov.u32 s4, %4;\n\t" \
//		"add.u32 t1, s1, s2;\n\t" \
//		"shf.r.clamp.b32 t2, s3, s3, 7;\n\t" \
//		"shf.r.clamp.b32 t3, s3, s3, 18;\n\t" \
//		"xor.b32 t2, t2, t3;\n\t" \
//		"shr.u32 t3, s3, 3;\n\t" \
//		"xor.b32 t2, t2 ,t3;\n\t" \
//		"add.u32 t1, t1, t2;\n\t" \
//		"shf.r.clamp.b32 t2, s4, s4, 17;\n\t" \
//		"shf.r.clamp.b32 t3, s4, s4, 19;\n\t" \
//		"xor.b32 t2, t2, t3;\n\t" \
//		"shr.u32 t3, %4, 10;\n\t" \
//		"xor.b32 t2, t2, t3;\n\t" \
//		"add.u32 t1, t1, t2;\n\t" \
//		"mov.u32 %0, t1;\n\t" \
//		"}" \
//		: "=r"(schedule[i]) : "r"(schedule[i - 16]), "r"(schedule[i - 7]), "r"(schedule[i - 15]), "r"(schedule[i - 2]));

//#define SCHEDULE(i) \
//	asm("{\n\t" \
//		".reg .u32 t2;\n\t" \
//		".reg .u32 t3;\n\t" \
//		"shf.r.clamp.b32 t2, %3, %3, 7;\n\t" \
//		"shf.r.clamp.b32 t3, %3, %3, 18;\n\t" \
//		"xor.b32 t2, t2, t3;\n\t" \
//		"shr.u32 t3, %3, 3;\n\t" \
//		"xor.b32 t2, t2 ,t3;\n\t" \
//		"add.u32 %0, %0, t2;\n\t" \
//		"shf.r.clamp.b32 t2, %4, %4, 17;\n\t" \
//		"shf.r.clamp.b32 t3, %4, %4, 19;\n\t" \
//		"xor.b32 t2, t2, t3;\n\t" \
//		"shr.u32 t3, %4, 10;\n\t" \
//		"xor.b32 t2, t2, t3;\n\t" \
//		"add.u32 %0, %0, t2;\n\t" \
//		"add.u32 t2, %1, %2;\n\t" \
//		"add.u32 %0, %0, t2;\n\t" \
//		"}" \
//		: "=r"(schedule[i]) : "r"(schedule[i - 16]), "r"(schedule[i - 7]), "r"(schedule[i - 15]), "r"(schedule[i - 2]));

#define ROUND(a, b, c, d, e, f, g, h, i, k) \
		h = 0U + h + (ROTR32(e, 6) ^ ROTR32(e, 11) ^ ROTR32(e, 25)) + (g ^ (e & (f ^ g))) + UINT32_C(k) + schedule[i];  \
		d = 0U + d + h;  \
		h = 0U + h + (ROTR32(a, 2) ^ ROTR32(a, 13) ^ ROTR32(a, 22)) + ((a & (b | c)) | (b & c));

#define CUDA_CALL(x) {const hipError_t a = (x);if(a!=hipSuccess){printf("\nCUDA Error:%s(err_num=%d)\n",hipGetErrorString(a),a);}}
#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \ printf("Error at %s:%d\n",__FILE__,__LINE__);\ return EXIT_FAILURE;}} while(0)

//__shared__ uint k[64];

__constant__ char constantAreaPlainCharSet[36];

__shared__ char plainCharSet[384][95];

__shared__ uint state[384][8];

struct ChainComparator {
	__host__ __device__
		bool operator()(const struct Chain& lhs, const struct Chain& rhs) {
		return lhs.indexE < rhs.indexE;
	}
};

struct HashCompartor {
	__host__ __device__
		bool operator()(const struct PasswordMapping& lhs, const struct PasswordMapping& rhs) {
		//const ulong* lhsP = (const ulong*)lhs.hash;
		//const ulong* rhsP = (const ulong*)rhs.hash;
		//ulong lhs4 = *(lhsP + 3);
		//ulong lhs3 = *(lhsP + 2);
		//ulong lhs2 = *(lhsP + 1);
		//ulong lhs1 = *(lhsP);
		//ulong rhs4 = *(rhsP + 3);
		//ulong rhs3 = *(rhsP + 2);
		//ulong rhs2 = *(rhsP + 1);
		//ulong rhs1 = *(rhsP);
		//return lhs1 < rhs1
		//	|| lhs1 == rhs1 && lhs2 < rhs2
		//	|| lhs1 == rhs1 && lhs2 == rhs2 && lhs3 < rhs3
		//	|| lhs1 == rhs1 && lhs2 == rhs2 && lhs3 == rhs3 && lhs4 < rhs4;
		bool flap = true;
		for (int i = 0; i < 32; i++) {
			if (lhs.hash[i] > rhs.hash[i]) {
				flap = false;
				break;
			}
		}
		if (flap) {
			return true;
		}
		else {
			return false;
		}
	}
};

void QSort(struct PasswordMapping* mappings, uint32_t CHAINS_SIZE) {
	thrust::device_ptr<struct PasswordMapping> thrustChainP(mappings);
	thrust::sort(thrustChainP, thrustChainP + CHAINS_SIZE, HashCompartor());
}

__device__ void indexToPlain(ulong index, const uint8_t plainLength,
	const uint8_t plainCharsetSize, char* plain)
{
	for (int i = plainLength - 1;i >= 0;i--) {
		plain[i] = index % plainCharsetSize;
		index /= plainCharsetSize;
	}
}

__device__ inline void indexToPlain(ulong index, char* plain, const uint8_t plainLength, const char* charSet, const unsigned int charSetSize)
{
	for (int i = plainLength - 1; i >= 0; i--) {
		plain[i] = charSet[(index & 0x7f) % charSetSize];
		index >>= 7;
	}
}

/*__device__ ulong plainToIndex(const char* plain, size_t plainLength, const char* charSet, size_t charSetSize, map<char, size_t>* charIndexMap)
{
ulong index = 0;
int i;

for (i = 0;i<plainLength - 1;i++) {
index += charIndexMap->operator[](plain[i]) & 0x7f;
index <<= 7;
}
index += charIndexMap->operator[](plain[i]) & 0x7f;
return index;
}*/

__device__ inline ulong hashToIndexPaperVersion(unsigned char* hash, int pos, const uint8_t plainCharSetSize)
{
	unsigned int* hashP = (unsigned int*)hash;
	unsigned int p0 = *(hashP + 4) ^ pos;
	unsigned int p2 = *(hashP + 5) ^ (pos >> 12);
	unsigned int p4 = *(hashP + 6) ^ (pos >> 24);
	unsigned int p6 = *(hashP + 7);
	char* plainCharSetP = plainCharSet[threadIdx.x];

	unsigned __int16 b0 = plainCharSetP[p0 % plainCharSetSize] << 8 | plainCharSetP[(p0 >> 16) % plainCharSetSize];
	unsigned __int16 b1 = plainCharSetP[p2 % plainCharSetSize] << 8 | plainCharSetP[(p2 >> 16) % plainCharSetSize];
	unsigned __int16 b2 = plainCharSetP[p4 % plainCharSetSize] << 8 | plainCharSetP[(p4 >> 16) % plainCharSetSize];
	unsigned __int16 b3 = plainCharSetP[p6 % plainCharSetSize] << 8 | plainCharSetP[(p6 >> 16) % plainCharSetSize];
	/*unsigned __int16 b0 = constantAreaPlainCharSet[p0 % plainCharSetSize] << 8 | constantAreaPlainCharSet[(p0 >> 16) % plainCharSetSize];
	unsigned __int16 b1 = constantAreaPlainCharSet[p2 % plainCharSetSize] << 8 | constantAreaPlainCharSet[(p2 >> 16) % plainCharSetSize];
	unsigned __int16 b2 = constantAreaPlainCharSet[p4 % plainCharSetSize] << 8 | constantAreaPlainCharSet[(p4 >> 16) % plainCharSetSize];
	unsigned __int16 b3 = constantAreaPlainCharSet[p6 % plainCharSetSize] << 8 | constantAreaPlainCharSet[(p6 >> 16) % plainCharSetSize];*/
	ulong index = 0;
	index += b0;
	index <<= 16;
	index += b1;
	index <<= 16;
	index += b2;
	index <<= 16;
	index += b3;
	return index;
}

__device__ inline ulong hashToIndexWithoutCharSet(unsigned char* hash, int pos, const uint8_t plainCharSetSize)
{
	unsigned int* hashP = (unsigned int*)hash;
	unsigned int p0 = *(hashP + 4) ^ pos;
	unsigned int p2 = *(hashP + 5) ^ (pos >> 12);
	unsigned int p4 = *(hashP + 6) ^ (pos >> 24);
	unsigned int p6 = *(hashP + 7);

	unsigned __int16 b0 = ((p0 % plainCharSetSize) << 8) | ((p0 >> 16) % plainCharSetSize);
	unsigned __int16 b1 = ((p2 % plainCharSetSize) << 8) | ((p2 >> 16) % plainCharSetSize);
	unsigned __int16 b2 = ((p4 % plainCharSetSize) << 8) | ((p4 >> 16) % plainCharSetSize);
	unsigned __int16 b3 = ((p6 % plainCharSetSize) << 8) | ((p6 >> 16) % plainCharSetSize);

	ulong index = 0;
	index += b0;
	index <<= 16;
	index += b1;
	index <<= 16;
	index += b2;
	index <<= 16;
	index += b3;
	return index;
}

__device__ inline void plainToHashWithInlinePTX(const char* plain, const uint8_t length, unsigned char* res) {
	unsigned int bitlen0 = 0;
	unsigned int bitlen1 = 0;
	unsigned int stateP[8];

	//unsigned int* stateP = state[threadIdx.x];

	unsigned char data[64];

	unsigned int l;

	for (l = 0; l < length; ++l) {
		data[l] = plain[l];
	}


	stateP[0] = 0x6a09e667;
	stateP[1] = 0xbb67ae85;
	stateP[2] = 0x3c6ef372;
	stateP[3] = 0xa54ff53a;
	stateP[4] = 0x510e527f;
	stateP[5] = 0x9b05688c;
	stateP[6] = 0x1f83d9ab;
	stateP[7] = 0x5be0cd19;


	data[l++] = 0x80;
	while (l < 56)
		data[l++] = 0x00;


	//// Append to the padding the total message's length in bits and transform. 
	DBL_INT_ADD(bitlen0, bitlen1, length * 8);

	data[63] = bitlen0;
	data[62] = bitlen0 >> 8;
	data[61] = bitlen0 >> 16;
	data[60] = bitlen0 >> 24;
	data[59] = bitlen1;
	data[58] = bitlen1 >> 8;
	data[57] = bitlen1 >> 16;
	data[56] = bitlen1 >> 24;

	uint32_t schedule[16];

	uint32_t a = stateP[0];
	uint32_t b = stateP[1];
	uint32_t c = stateP[2];
	uint32_t d = stateP[3];
	uint32_t e = stateP[4];
	uint32_t f = stateP[5];
	uint32_t g = stateP[6];
	uint32_t h = stateP[7];

	ROUNDa(0, a, b, c, d, e, f, g, h, 0x428A2F98)
	ROUNDa(1, h, a, b, c, d, e, f, g, 0x71374491)
	ROUNDa(2, g, h, a, b, c, d, e, f, -0x4A3F0431)
	ROUNDa(3, f, g, h, a, b, c, d, e, -0x164A245B)
	ROUNDa(4, e, f, g, h, a, b, c, d, 0x3956C25B)
	ROUNDa(5, d, e, f, g, h, a, b, c, 0x59F111F1)
	ROUNDa(6, c, d, e, f, g, h, a, b, -0x6DC07D5C)
	ROUNDa(7, b, c, d, e, f, g, h, a, -0x54E3A12B)
	ROUNDa(8, a, b, c, d, e, f, g, h, -0x27F85568)
	ROUNDa(9, h, a, b, c, d, e, f, g, 0x12835B01)
	ROUNDa(10, g, h, a, b, c, d, e, f, 0x243185BE)
	ROUNDa(11, f, g, h, a, b, c, d, e, 0x550C7DC3)
	ROUNDa(12, e, f, g, h, a, b, c, d, 0x72BE5D74)
	ROUNDa(13, d, e, f, g, h, a, b, c, -0x7F214E02)
	ROUNDa(14, c, d, e, f, g, h, a, b, -0x6423F959)
	ROUNDa(15, b, c, d, e, f, g, h, a, -0x3E640E8C)
	ROUND16(16, a, b, c, d, e, f, g, h, -0x1B64963F)
	ROUND17(17, h, a, b, c, d, e, f, g, -0x1041B87A)
	ROUND18(18, g, h, a, b, c, d, e, f, 0x0FC19DC6)
	ROUND19(19, f, g, h, a, b, c, d, e, 0x240CA1CC)
	ROUND20(20, e, f, g, h, a, b, c, d, 0x2DE92C6F)
	ROUND21(21, d, e, f, g, h, a, b, c, 0x4A7484AA)
	ROUND22(22, c, d, e, f, g, h, a, b, 0x5CB0A9DC)
	ROUND23(23, b, c, d, e, f, g, h, a, 0x76F988DA)
	ROUND24(24, a, b, c, d, e, f, g, h, -0x67C1AEAE)
	ROUND25(25, h, a, b, c, d, e, f, g, -0x57CE3993)
	ROUND26(26, g, h, a, b, c, d, e, f, -0x4FFCD838)
	ROUND27(27, f, g, h, a, b, c, d, e, -0x40A68039)
	ROUND28(28, e, f, g, h, a, b, c, d, -0x391FF40D)
	ROUND29(29, d, e, f, g, h, a, b, c, -0x2A586EB9)
	ROUND30(30, c, d, e, f, g, h, a, b, 0x06CA6351)
	ROUND31(31, b, c, d, e, f, g, h, a, 0x14292967)
	ROUND16(32, a, b, c, d, e, f, g, h, 0x27B70A85)
	ROUND17(33, h, a, b, c, d, e, f, g, 0x2E1B2138)
	ROUND18(34, g, h, a, b, c, d, e, f, 0x4D2C6DFC)
	ROUND19(35, f, g, h, a, b, c, d, e, 0x53380D13)
	ROUND20(36, e, f, g, h, a, b, c, d, 0x650A7354)
	ROUND21(37, d, e, f, g, h, a, b, c, 0x766A0ABB)
	ROUND22(38, c, d, e, f, g, h, a, b, -0x7E3D36D2)
	ROUND23(39, b, c, d, e, f, g, h, a, -0x6D8DD37B)
	ROUND24(40, a, b, c, d, e, f, g, h, -0x5D40175F)
	ROUND25(41, h, a, b, c, d, e, f, g, -0x57E599B5)
	ROUND26(42, g, h, a, b, c, d, e, f, -0x3DB47490)
	ROUND27(43, f, g, h, a, b, c, d, e, -0x3893AE5D)
	ROUND28(44, e, f, g, h, a, b, c, d, -0x2E6D17E7)
	ROUND29(45, d, e, f, g, h, a, b, c, -0x2966F9DC)
	ROUND30(46, c, d, e, f, g, h, a, b, -0x0BF1CA7B)
	ROUND31(47, b, c, d, e, f, g, h, a, 0x106AA070)
	ROUND16(48, a, b, c, d, e, f, g, h, 0x19A4C116)
	ROUND17(49, h, a, b, c, d, e, f, g, 0x1E376C08)
	ROUND18(50, g, h, a, b, c, d, e, f, 0x2748774C)
	ROUND19(51, f, g, h, a, b, c, d, e, 0x34B0BCB5)
	ROUND20(52, e, f, g, h, a, b, c, d, 0x391C0CB3)
	ROUND21(53, d, e, f, g, h, a, b, c, 0x4ED8AA4A)
	ROUND22(54, c, d, e, f, g, h, a, b, 0x5B9CCA4F)
	ROUND23(55, b, c, d, e, f, g, h, a, 0x682E6FF3)
	ROUND24(56, a, b, c, d, e, f, g, h, 0x748F82EE)
	ROUND25(57, h, a, b, c, d, e, f, g, 0x78A5636F)
	ROUND26(58, g, h, a, b, c, d, e, f, -0x7B3787EC)
	ROUND27(59, f, g, h, a, b, c, d, e, -0x7338FDF8)
	ROUND28(60, e, f, g, h, a, b, c, d, -0x6F410006)
	ROUND29(61, d, e, f, g, h, a, b, c, -0x5BAF9315)
	ROUND30(62, c, d, e, f, g, h, a, b, -0x41065C09)
	ROUND31(63, b, c, d, e, f, g, h, a, -0x398E870E)

	stateP[0] += a;
	stateP[1] += b;
	stateP[2] += c;
	stateP[3] += d;
	stateP[4] += e;
	stateP[5] += f;
	stateP[6] += g;
	stateP[7] += h;

	// Since this implementation uses little endian byte ordering and SHA uses big endian,
	// reverse all the bytes when copying the final state to the output hash. 

	for (int i = 0; i < 4; ++i) {
		l = i << 3;
		*(res) = (stateP[0] >> (24 - l)) & 0x000000ff;
		*(res + 4) = (stateP[1] >> (24 - l)) & 0x000000ff;
		*(res + 8) = (stateP[2] >> (24 - l)) & 0x000000ff;
		*(res + 12) = (stateP[3] >> (24 - l)) & 0x000000ff;
		*(res + 16) = (stateP[4] >> (24 - l)) & 0x000000ff;
		*(res + 20) = (stateP[5] >> (24 - l)) & 0x000000ff;
		*(res + 24) = (stateP[6] >> (24 - l)) & 0x000000ff;
		*(res + 28) = (stateP[7] >> (24 - l)) & 0x000000ff;
		++res;
	}

	//unsigned int* resP = (unsigned int*)res;
	//unsigned char* stateCP = (unsigned char*)stateP;

	//*(resP) = (((unsigned int)*(stateCP)<<0)| ((unsigned int)*(stateCP+1)<<8)| ((unsigned int)*(stateCP+2)<<16)| ((unsigned int)*(stateCP+3)<<24));
	//*(resP+1) = ((unsigned int)(*(stateCP+4) << 0) | ((unsigned int)*(stateCP + 5) << 8) | ((unsigned int)*(stateCP + 6) << 16) | ((unsigned int)*(stateCP + 7) << 24));
	//*(resP+2) = (((unsigned int)*(stateCP+8) << 0) | ((unsigned int)*(stateCP + 9) << 8) | ((unsigned int)*(stateCP + 10) << 16) | ((unsigned int)*(stateCP + 11) << 24));
	//*(resP+3) = (((unsigned int)*(stateCP+12) << 0) | ((unsigned int)*(stateCP + 13) << 8) | ((unsigned int)*(stateCP + 14) << 16) | ((unsigned int)*(stateCP + 15) << 24));
	//*(resP+4) = (((unsigned int)*(stateCP+16) << 0) | ((unsigned int)*(stateCP + 17) << 8) | ((unsigned int)*(stateCP + 18) << 16) | ((unsigned int)*(stateCP + 19) << 24));
	//*(resP+5) = (((unsigned int)*(stateCP+20) << 0) | ((unsigned int)*(stateCP + 21) << 8) | ((unsigned int)*(stateCP + 22) << 16) | ((unsigned int)*(stateCP + 23) << 24));
	//*(resP+6) = (((unsigned int)*(stateCP+24) << 0) | ((unsigned int)*(stateCP + 25) << 8) | ((unsigned int)*(stateCP + 26) << 16) | ((unsigned int)*(stateCP + 27) << 24));
	//*(resP+7) = (((unsigned int)*(stateCP+28) << 0) | ((unsigned int)*(stateCP + 29) << 8) | ((unsigned int)*(stateCP + 30) << 16) | ((unsigned int)*(stateCP + 31) << 24));

	///**((unsigned int*)res) = ((*((unsigned char*)stateP) << 0) | (*((unsigned char*)stateP + 1) << 8) | (*((unsigned char*)stateP + 2) << 16) | (*((unsigned char*)stateP + 3) << 24));
	//*((unsigned int*)res + 1) = ((*((unsigned char*)stateP + 4) << 0) | (*((unsigned char*)stateP + 5) << 8) | (*((unsigned char*)stateP + 6) << 16) | (*((unsigned char*)stateP + 7) << 24));
	//*((unsigned int*)res + 2) = ((*((unsigned char*)stateP + 8) << 0) | (*((unsigned char*)stateP + 9) << 8) | (*((unsigned char*)stateP + 10) << 16) | (*((unsigned char*)stateP + 11) << 24));
	//*((unsigned int*)res + 3) = ((*((unsigned char*)stateP + 12) << 0) | (*((unsigned char*)stateP + 13) << 8) | (*((unsigned char*)stateP + 14) << 16) | (*((unsigned char*)stateP + 15) << 24));
	//*((unsigned int*)res + 4) = ((*((unsigned char*)stateP + 16) << 0) | (*((unsigned char*)stateP + 17) << 8) | (*((unsigned char*)stateP + 18) << 16) | (*((unsigned char*)stateP + 19) << 24));
	//*((unsigned int*)res + 5) = ((*((unsigned char*)stateP + 20) << 0) | (*((unsigned char*)stateP + 21) << 8) | (*((unsigned char*)stateP + 22) << 16) | (*((unsigned char*)stateP + 23) << 24));
	//*((unsigned int*)res + 6) = ((*((unsigned char*)stateP + 24) << 0) | (*((unsigned char*)stateP + 25) << 8) | (*((unsigned char*)stateP + 26) << 16) | (*((unsigned char*)stateP + 27) << 24));
	//*((unsigned int*)res + 7) = ((*((unsigned char*)stateP + 28) << 0) | (*((unsigned char*)stateP + 29) << 8) | (*((unsigned char*)stateP + 30) << 16) | (*((unsigned char*)stateP + 31) << 24));*/
}

__device__ inline void plainToHashWithInlinePTX(ulong index, const uint8_t length, unsigned char* res, const uint8_t charSetSize) {
	unsigned int bitlen0 = 0;
	unsigned int bitlen1 = 0;
	unsigned int stateP[8];

	unsigned char data[64];

	unsigned int l;

	// reduct the index in the plain space
	for (l = length - 1; l >= 1; l--) {
		data[l] = (index & 0x7f) % charSetSize + 32;
		index >>= 7;
	}
	data[0] = (index & 0x7f) % charSetSize + 32;
	l = length;

	stateP[0] = 0x6a09e667;
	stateP[1] = 0xbb67ae85;
	stateP[2] = 0x3c6ef372;
	stateP[3] = 0xa54ff53a;
	stateP[4] = 0x510e527f;
	stateP[5] = 0x9b05688c;
	stateP[6] = 0x1f83d9ab;
	stateP[7] = 0x5be0cd19;


	data[l++] = 0x80;
	while (l < 56)
		data[l++] = 0x00;


	//// Append to the padding the total message's length in bits and transform. 
	DBL_INT_ADD(bitlen0, bitlen1, length * 8);

	data[63] = bitlen0;
	data[62] = bitlen0 >> 8;
	data[61] = bitlen0 >> 16;
	data[60] = bitlen0 >> 24;
	data[59] = bitlen1;
	data[58] = bitlen1 >> 8;
	data[57] = bitlen1 >> 16;
	data[56] = bitlen1 >> 24;

	uint32_t schedule[16];

	uint32_t a = stateP[0];
	uint32_t b = stateP[1];
	uint32_t c = stateP[2];
	uint32_t d = stateP[3];
	uint32_t e = stateP[4];
	uint32_t f = stateP[5];
	uint32_t g = stateP[6];
	uint32_t h = stateP[7];

	ROUNDa(0, a, b, c, d, e, f, g, h, 0x428A2F98)
	ROUNDa(1, h, a, b, c, d, e, f, g, 0x71374491)
	ROUNDa(2, g, h, a, b, c, d, e, f, -0x4A3F0431)
	ROUNDa(3, f, g, h, a, b, c, d, e, -0x164A245B)
	ROUNDa(4, e, f, g, h, a, b, c, d, 0x3956C25B)
	ROUNDa(5, d, e, f, g, h, a, b, c, 0x59F111F1)
	ROUNDa(6, c, d, e, f, g, h, a, b, -0x6DC07D5C)
	ROUNDa(7, b, c, d, e, f, g, h, a, -0x54E3A12B)
	ROUNDa(8, a, b, c, d, e, f, g, h, -0x27F85568)
	ROUNDa(9, h, a, b, c, d, e, f, g, 0x12835B01)
	ROUNDa(10, g, h, a, b, c, d, e, f, 0x243185BE)
	ROUNDa(11, f, g, h, a, b, c, d, e, 0x550C7DC3)
	ROUNDa(12, e, f, g, h, a, b, c, d, 0x72BE5D74)
	ROUNDa(13, d, e, f, g, h, a, b, c, -0x7F214E02)
	ROUNDa(14, c, d, e, f, g, h, a, b, -0x6423F959)
	ROUNDa(15, b, c, d, e, f, g, h, a, -0x3E640E8C)
	ROUND16(16, a, b, c, d, e, f, g, h, -0x1B64963F)
	ROUND17(17, h, a, b, c, d, e, f, g, -0x1041B87A)
	ROUND18(18, g, h, a, b, c, d, e, f, 0x0FC19DC6)
	ROUND19(19, f, g, h, a, b, c, d, e, 0x240CA1CC)
	ROUND20(20, e, f, g, h, a, b, c, d, 0x2DE92C6F)
	ROUND21(21, d, e, f, g, h, a, b, c, 0x4A7484AA)
	ROUND22(22, c, d, e, f, g, h, a, b, 0x5CB0A9DC)
	ROUND23(23, b, c, d, e, f, g, h, a, 0x76F988DA)
	ROUND24(24, a, b, c, d, e, f, g, h, -0x67C1AEAE)
	ROUND25(25, h, a, b, c, d, e, f, g, -0x57CE3993)
	ROUND26(26, g, h, a, b, c, d, e, f, -0x4FFCD838)
	ROUND27(27, f, g, h, a, b, c, d, e, -0x40A68039)
	ROUND28(28, e, f, g, h, a, b, c, d, -0x391FF40D)
	ROUND29(29, d, e, f, g, h, a, b, c, -0x2A586EB9)
	ROUND30(30, c, d, e, f, g, h, a, b, 0x06CA6351)
	ROUND31(31, b, c, d, e, f, g, h, a, 0x14292967)
	ROUND16(32, a, b, c, d, e, f, g, h, 0x27B70A85)
	ROUND17(33, h, a, b, c, d, e, f, g, 0x2E1B2138)
	ROUND18(34, g, h, a, b, c, d, e, f, 0x4D2C6DFC)
	ROUND19(35, f, g, h, a, b, c, d, e, 0x53380D13)
	ROUND20(36, e, f, g, h, a, b, c, d, 0x650A7354)
	ROUND21(37, d, e, f, g, h, a, b, c, 0x766A0ABB)
	ROUND22(38, c, d, e, f, g, h, a, b, -0x7E3D36D2)
	ROUND23(39, b, c, d, e, f, g, h, a, -0x6D8DD37B)
	ROUND24(40, a, b, c, d, e, f, g, h, -0x5D40175F)
	ROUND25(41, h, a, b, c, d, e, f, g, -0x57E599B5)
	ROUND26(42, g, h, a, b, c, d, e, f, -0x3DB47490)
	ROUND27(43, f, g, h, a, b, c, d, e, -0x3893AE5D)
	ROUND28(44, e, f, g, h, a, b, c, d, -0x2E6D17E7)
	ROUND29(45, d, e, f, g, h, a, b, c, -0x2966F9DC)
	ROUND30(46, c, d, e, f, g, h, a, b, -0x0BF1CA7B)
	ROUND31(47, b, c, d, e, f, g, h, a, 0x106AA070)
	ROUND16(48, a, b, c, d, e, f, g, h, 0x19A4C116)
	ROUND17(49, h, a, b, c, d, e, f, g, 0x1E376C08)
	ROUND18(50, g, h, a, b, c, d, e, f, 0x2748774C)
	ROUND19(51, f, g, h, a, b, c, d, e, 0x34B0BCB5)
	ROUND20(52, e, f, g, h, a, b, c, d, 0x391C0CB3)
	ROUND21(53, d, e, f, g, h, a, b, c, 0x4ED8AA4A)
	ROUND22(54, c, d, e, f, g, h, a, b, 0x5B9CCA4F)
	ROUND23(55, b, c, d, e, f, g, h, a, 0x682E6FF3)
	ROUND24(56, a, b, c, d, e, f, g, h, 0x748F82EE)
	ROUND25(57, h, a, b, c, d, e, f, g, 0x78A5636F)
	ROUND26(58, g, h, a, b, c, d, e, f, -0x7B3787EC)
	ROUND27(59, f, g, h, a, b, c, d, e, -0x7338FDF8)
	ROUND28(60, e, f, g, h, a, b, c, d, -0x6F410006)
	ROUND29(61, d, e, f, g, h, a, b, c, -0x5BAF9315)
	ROUND30(62, c, d, e, f, g, h, a, b, -0x41065C09)
	ROUND31(63, b, c, d, e, f, g, h, a, -0x398E870E)

	stateP[0] += a;
	stateP[1] += b;
	stateP[2] += c;
	stateP[3] += d;
	stateP[4] += e;
	stateP[5] += f;
	stateP[6] += g;
	stateP[7] += h;

	// Since this implementation uses little endian byte ordering and SHA uses big endian,
	// reverse all the bytes when copying the final state to the output hash. 

	for (int i = 0; i < 4; ++i) {
		l = i << 3;
		*(res) = (stateP[0] >> (24 - l)) & 0x000000ff;
		*(res + 4) = (stateP[1] >> (24 - l)) & 0x000000ff;
		*(res + 8) = (stateP[2] >> (24 - l)) & 0x000000ff;
		*(res + 12) = (stateP[3] >> (24 - l)) & 0x000000ff;
		*(res + 16) = (stateP[4] >> (24 - l)) & 0x000000ff;
		*(res + 20) = (stateP[5] >> (24 - l)) & 0x000000ff;
		*(res + 24) = (stateP[6] >> (24 - l)) & 0x000000ff;
		*(res + 28) = (stateP[7] >> (24 - l)) & 0x000000ff;
		++res;
	}
}

__device__ inline void plainToHash(char* plain, const uint8_t length, unsigned char* res)
{
	unsigned int bitlen0 = 0;
	unsigned int bitlen1 = 0;
	//unsigned int stateP[8];

	unsigned char data[64];

	unsigned int l;

	for (l = 0; l < length; ++l) {
		data[l] = plain[l];
	}

	uint* stateP = state[threadIdx.x];

	stateP[0] = 0x6a09e667;
	stateP[1] = 0xbb67ae85;
	stateP[2] = 0x3c6ef372;
	stateP[3] = 0xa54ff53a;
	stateP[4] = 0x510e527f;
	stateP[5] = 0x9b05688c;
	stateP[6] = 0x1f83d9ab;
	stateP[7] = 0x5be0cd19;


	// Pad whatever data is left in the buffer. 
	data[l++] = 0x80;
	while (l < 56)
		data[l++] = 0x00;


	// Append to the padding the total message's length in bits and transform. 
	DBL_INT_ADD(bitlen0, bitlen1, length * 8);
	data[63] = bitlen0;
	data[62] = bitlen0 >> 8;
	data[61] = bitlen0 >> 16;
	data[60] = bitlen0 >> 24;
	data[59] = bitlen1;
	data[58] = bitlen1 >> 8;
	data[57] = bitlen1 >> 16;
	data[56] = bitlen1 >> 24;


	uint32_t schedule[64];
	LOADSCHEDULE(0)
	LOADSCHEDULE(1)
	LOADSCHEDULE(2)
	LOADSCHEDULE(3)
	LOADSCHEDULE(4)
	LOADSCHEDULE(5)
	LOADSCHEDULE(6)
	LOADSCHEDULE(7)
	LOADSCHEDULE(8)
	LOADSCHEDULE(9)
	LOADSCHEDULE(10)
	LOADSCHEDULE(11)
	LOADSCHEDULE(12)
	LOADSCHEDULE(13)
	LOADSCHEDULE(14)
	LOADSCHEDULE(15)
	SCHEDULE(16)
	SCHEDULE(17)
	SCHEDULE(18)
	SCHEDULE(19)
	SCHEDULE(20)
	SCHEDULE(21)
	SCHEDULE(22)
	SCHEDULE(23)
	SCHEDULE(24)
	SCHEDULE(25)
	SCHEDULE(26)
	SCHEDULE(27)
	SCHEDULE(28)
	SCHEDULE(29)
	SCHEDULE(30)
	SCHEDULE(31)
	SCHEDULE(32)
	SCHEDULE(33)
	SCHEDULE(34)
	SCHEDULE(35)
	SCHEDULE(36)
	SCHEDULE(37)
	SCHEDULE(38)
	SCHEDULE(39)
	SCHEDULE(40)
	SCHEDULE(41)
	SCHEDULE(42)
	SCHEDULE(43)
	SCHEDULE(44)
	SCHEDULE(45)
	SCHEDULE(46)
	SCHEDULE(47)
	SCHEDULE(48)
	SCHEDULE(49)
	SCHEDULE(50)
	SCHEDULE(51)
	SCHEDULE(52)
	SCHEDULE(53)
	SCHEDULE(54)
	SCHEDULE(55)
	SCHEDULE(56)
	SCHEDULE(57)
	SCHEDULE(58)
	SCHEDULE(59)
	SCHEDULE(60)
	SCHEDULE(61)
	SCHEDULE(62)
	SCHEDULE(63)

	uint32_t a = stateP[0];
	uint32_t b = stateP[1];
	uint32_t c = stateP[2];
	uint32_t d = stateP[3];
	uint32_t e = stateP[4];
	uint32_t f = stateP[5];
	uint32_t g = stateP[6];
	uint32_t h = stateP[7];
	ROUND(a, b, c, d, e, f, g, h, 0, 0x428A2F98)
	ROUND(h, a, b, c, d, e, f, g, 1, 0x71374491)
	ROUND(g, h, a, b, c, d, e, f, 2, 0xB5C0FBCF)
	ROUND(f, g, h, a, b, c, d, e, 3, 0xE9B5DBA5)
	ROUND(e, f, g, h, a, b, c, d, 4, 0x3956C25B)
	ROUND(d, e, f, g, h, a, b, c, 5, 0x59F111F1)
	ROUND(c, d, e, f, g, h, a, b, 6, 0x923F82A4)
	ROUND(b, c, d, e, f, g, h, a, 7, 0xAB1C5ED5)
	ROUND(a, b, c, d, e, f, g, h, 8, 0xD807AA98)
	ROUND(h, a, b, c, d, e, f, g, 9, 0x12835B01)
	ROUND(g, h, a, b, c, d, e, f, 10, 0x243185BE)
	ROUND(f, g, h, a, b, c, d, e, 11, 0x550C7DC3)
	ROUND(e, f, g, h, a, b, c, d, 12, 0x72BE5D74)
	ROUND(d, e, f, g, h, a, b, c, 13, 0x80DEB1FE)
	ROUND(c, d, e, f, g, h, a, b, 14, 0x9BDC06A7)
	ROUND(b, c, d, e, f, g, h, a, 15, 0xC19BF174)
	ROUND(a, b, c, d, e, f, g, h, 16, 0xE49B69C1)
	ROUND(h, a, b, c, d, e, f, g, 17, 0xEFBE4786)
	ROUND(g, h, a, b, c, d, e, f, 18, 0x0FC19DC6)
	ROUND(f, g, h, a, b, c, d, e, 19, 0x240CA1CC)
	ROUND(e, f, g, h, a, b, c, d, 20, 0x2DE92C6F)
	ROUND(d, e, f, g, h, a, b, c, 21, 0x4A7484AA)
	ROUND(c, d, e, f, g, h, a, b, 22, 0x5CB0A9DC)
	ROUND(b, c, d, e, f, g, h, a, 23, 0x76F988DA)
	ROUND(a, b, c, d, e, f, g, h, 24, 0x983E5152)
	ROUND(h, a, b, c, d, e, f, g, 25, 0xA831C66D)
	ROUND(g, h, a, b, c, d, e, f, 26, 0xB00327C8)
	ROUND(f, g, h, a, b, c, d, e, 27, 0xBF597FC7)
	ROUND(e, f, g, h, a, b, c, d, 28, 0xC6E00BF3)
	ROUND(d, e, f, g, h, a, b, c, 29, 0xD5A79147)
	ROUND(c, d, e, f, g, h, a, b, 30, 0x06CA6351)
	ROUND(b, c, d, e, f, g, h, a, 31, 0x14292967)
	ROUND(a, b, c, d, e, f, g, h, 32, 0x27B70A85)
	ROUND(h, a, b, c, d, e, f, g, 33, 0x2E1B2138)
	ROUND(g, h, a, b, c, d, e, f, 34, 0x4D2C6DFC)
	ROUND(f, g, h, a, b, c, d, e, 35, 0x53380D13)
	ROUND(e, f, g, h, a, b, c, d, 36, 0x650A7354)
	ROUND(d, e, f, g, h, a, b, c, 37, 0x766A0ABB)
	ROUND(c, d, e, f, g, h, a, b, 38, 0x81C2C92E)
	ROUND(b, c, d, e, f, g, h, a, 39, 0x92722C85)
	ROUND(a, b, c, d, e, f, g, h, 40, 0xA2BFE8A1)
	ROUND(h, a, b, c, d, e, f, g, 41, 0xA81A664B)
	ROUND(g, h, a, b, c, d, e, f, 42, 0xC24B8B70)
	ROUND(f, g, h, a, b, c, d, e, 43, 0xC76C51A3)
	ROUND(e, f, g, h, a, b, c, d, 44, 0xD192E819)
	ROUND(d, e, f, g, h, a, b, c, 45, 0xD6990624)
	ROUND(c, d, e, f, g, h, a, b, 46, 0xF40E3585)
	ROUND(b, c, d, e, f, g, h, a, 47, 0x106AA070)
	ROUND(a, b, c, d, e, f, g, h, 48, 0x19A4C116)
	ROUND(h, a, b, c, d, e, f, g, 49, 0x1E376C08)
	ROUND(g, h, a, b, c, d, e, f, 50, 0x2748774C)
	ROUND(f, g, h, a, b, c, d, e, 51, 0x34B0BCB5)
	ROUND(e, f, g, h, a, b, c, d, 52, 0x391C0CB3)
	ROUND(d, e, f, g, h, a, b, c, 53, 0x4ED8AA4A)
	ROUND(c, d, e, f, g, h, a, b, 54, 0x5B9CCA4F)
	ROUND(b, c, d, e, f, g, h, a, 55, 0x682E6FF3)
	ROUND(a, b, c, d, e, f, g, h, 56, 0x748F82EE)
	ROUND(h, a, b, c, d, e, f, g, 57, 0x78A5636F)
	ROUND(g, h, a, b, c, d, e, f, 58, 0x84C87814)
	ROUND(f, g, h, a, b, c, d, e, 59, 0x8CC70208)
	ROUND(e, f, g, h, a, b, c, d, 60, 0x90BEFFFA)
	ROUND(d, e, f, g, h, a, b, c, 61, 0xA4506CEB)
	ROUND(c, d, e, f, g, h, a, b, 62, 0xBEF9A3F7)
	ROUND(b, c, d, e, f, g, h, a, 63, 0xC67178F2)
	stateP[0] += a;
	stateP[1] += b;
	stateP[2] += c;
	stateP[3] += d;
	stateP[4] += e;
	stateP[5] += f;
	stateP[6] += g;
	stateP[7] += h;

	// Since this implementation uses little endian byte ordering and SHA uses big endian,
	// reverse all the bytes when copying the final state to the output hash. 

	for (int i = 0; i < 4; ++i) {
		l = i << 3;
		*(res) = (stateP[0] >> (24 - l)) & 0x000000ff;
		*(res + 4) = (stateP[1] >> (24 - l)) & 0x000000ff;
		*(res + 8) = (stateP[2] >> (24 - l)) & 0x000000ff;
		*(res + 12) = (stateP[3] >> (24 - l)) & 0x000000ff;
		*(res + 16) = (stateP[4] >> (24 - l)) & 0x000000ff;
		*(res + 20) = (stateP[5] >> (24 - l)) & 0x000000ff;
		*(res + 24) = (stateP[6] >> (24 - l)) & 0x000000ff;
		*(res + 28) = (stateP[7] >> (24 - l)) & 0x000000ff;
		++res;
	}
}

__device__ inline void initSHA256ConstantAndCharSet(const unsigned int charSetSize)
{
	char* plainCharSetP = plainCharSet[threadIdx.x];
	//for (i = 0;i < charSetSize;i++) {
	//	plainCharSetP[i] = srcCharSet[i];
	//}
	for (int i = 0;i < charSetSize;i++) {
		plainCharSetP[i] = constantAreaPlainCharSet[i];
	}
}

__device__ inline ulong hashToIndex(unsigned char* hash, int pos, ulong plainSpace)
{
	ulong* hashP = (ulong*)hash;
	return (ulong)((*(hashP)+ pos)) % plainSpace;
}

__device__ ulong hashToIndex(unsigned char* hash, int pos)
{
	ulong* hashP = (ulong*)hash;
	return (ulong)(((*(hashP) ^ *(hashP + 1) ^ *(hashP + 2) ^ *(hashP + 3)) + pos));
}

__device__ inline ulong reductFinalIndex(ulong index, uint8_t plainLength, uint8_t plainCharSize)
{
	ulong res = 0;
	uint8_t plainIndex[9];
	for (int l = plainLength - 1; l >= 0; l--) {
		// 32 - 126
		plainIndex[l] = ((uint8_t)(index & 0x7f)) % plainCharSize + 32;
		index >>= 7;
	}
	int j;
	for (j = 0; j < plainLength - 1; j++) {
		res += plainIndex[j];
		res <<= 7;
	}
	res += plainIndex[j];
	return res;
}

__global__ void generateChainPaperVersion(struct Chain* chains, const uint8_t plainCharSetSize,
	const uint8_t plainLength, const unsigned int chainLength)
{
	//initSHA256ConstantAndCharSet(plainCharSetSize);

	unsigned char hash[32];

	char plain[8];

	uint offset = (blockIdx.x * blockDim.x) + threadIdx.x;

	struct Chain* chain;

	chain = chains + offset;

	ulong indexE = chain->indexS;

	for (int i = 0;i < chainLength;i++) {
		//plainToHashWithInlinePTX((char *)&indexE, INDEX_SIZE_IN_BYTES, hash);
		plainToHashWithInlinePTX(indexE, plainLength, hash, plainCharSetSize);
		//indexE = hashToIndexWithoutCharSet(hash, i, plainCharSetSize);
		indexE = hashToIndex(hash, i);
	}
	chain->indexE = reductFinalIndex(indexE,plainLength,plainCharSetSize);
	

	//for (int i = 0;i < chainLength;i++) {
	//	indexToPlain(indexE, plainLength, plainCharSetSize, plain);
	//	plainToHashWithInlinePTX(plain, plainLength, hash);
	//	hashToIndex(hash, i);
	//}
}

__global__ void generateChainPaperVersion(struct Chain* chains, const uint8_t plainCharSetSize,
	const uint8_t plainLength, const unsigned int chainLength, ulong plainSpace)
{
	//initSHA256ConstantAndCharSet(plainCharSetSize);

	unsigned char hash[32];

	char plain[8];

	uint offset = (blockIdx.x * blockDim.x) + threadIdx.x;

	struct Chain* chain;

	chain = chains + offset;

	ulong indexE = chain->indexS;

	//for (int i = 0;i < chainLength;i++) {
	//	//plainToHashWithInlinePTX((char *)&indexE, INDEX_SIZE_IN_BYTES, hash);
	//	plainToHashWithInlinePTX(indexE, plainLength, hash, plainCharSetSize);
	//	//indexE = hashToIndexWithoutCharSet(hash, i, plainCharSetSize);
	//	indexE = hashToIndex(hash, i, 0x0fffffff);
	//}
	//chain->indexE = reductFinalIndex(indexE,plainLength,plainCharSetSize);
	//}

	for (int i = 0;i < chainLength;i++) {
		indexToPlain(indexE, plainLength, plainCharSetSize, plain);
		plainToHashWithInlinePTX(plain, plainLength, hash);
		indexE = hashToIndex(hash, i, plainSpace);
	}

	chain->indexE = indexE;
}

__global__ void generateChain(struct PasswordMapping* chains, const uint8_t plainCharSetSize)
{
	uint32_t offset = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (offset < plainCharSetSize) {
		struct PasswordMapping* chain = chains + offset;
		plainToHash(chain->plain, 1, chain->hash);
	}else if (offset < plainCharSetSize + plainCharSetSize * plainCharSetSize) {
		struct PasswordMapping* chain = chains + offset;
		plainToHash(chain->plain, 2, chain->hash);
	}else if (offset < plainCharSetSize + plainCharSetSize * plainCharSetSize + plainCharSetSize * plainCharSetSize * plainCharSetSize) {
		struct PasswordMapping* chain = chains + offset;
		plainToHash(chain->plain, 3, chain->hash);
	}
}

void generateTableWhilePasswordLengthLowerOrEqualThan3(const char* hostCharSetPath , const uint8_t plainCharSetSize)
{
	const uint32_t CHAINS_SIZE = plainCharSetSize + plainCharSetSize * plainCharSetSize + plainCharSetSize * plainCharSetSize * plainCharSetSize;
	struct PasswordMapping* deviceChains;
	struct PasswordMapping* hostChains;
	char* hostCharSet;
	CUDA_CALL(hipHostAlloc(&hostChains, CHAINS_SIZE * sizeof(struct PasswordMapping), hipHostMallocDefault));
	CUDA_CALL(hipHostAlloc(&hostCharSet, plainCharSetSize * sizeof(char), hipHostMallocDefault));
	getCharSet(hostCharSet, hostCharSetPath, plainCharSetSize);
	
	generateInitialIndex(hostChains, hostCharSet, plainCharSetSize);

	CUDA_CALL(hipMalloc(&deviceChains, CHAINS_SIZE * sizeof(struct PasswordMapping)));
	CUDA_CALL(hipMemcpy(deviceChains, hostChains, CHAINS_SIZE * sizeof(struct PasswordMapping), hipMemcpyHostToDevice));

	uint32_t threadPerBlock = 384;
	uint32_t blockNum = CHAINS_SIZE / threadPerBlock + 1;
	
	hipEvent_t startEvent;
	hipEvent_t endEvent;
	float cudaElapsedTime = 0.0f;
	hipEventCreate(&startEvent);
	hipEventCreate(&endEvent);
	hipEventRecord(startEvent, 0);

	generateChain<<<blockNum, threadPerBlock>>>(deviceChains, plainCharSetSize);

	hipEventRecord(endEvent, 0);
	hipEventSynchronize(endEvent);
	hipEventElapsedTime(&cudaElapsedTime, startEvent, endEvent);

	QSort(deviceChains, CHAINS_SIZE);

	CUDA_CALL(hipMemcpy(hostChains, deviceChains, CHAINS_SIZE * sizeof(struct PasswordMapping), hipMemcpyDeviceToHost));
	
	writeToFile((string("../") + "1-3#" + "ascii-32-95#" + "1").c_str(), hostChains, sizeof(struct PasswordMapping), CHAINS_SIZE);

	hipHostFree(hostChains);
	hipHostFree(hostCharSet);
	hipFree(deviceChains);
	//hipEventDestroy(startEvent);
	//hipEventDestroy(endEvent);

	hipDeviceReset();

	printf("%.3lf MH/S", (CHAINS_SIZE) / (cudaElapsedTime * 1000.0));
}

//int main()
//{
//	generateTableWhilePasswordLengthLowerOrEqualThan3("../charsets/ascii-32-95.txt", 95);
//
//	//constexpr uint32_t CHAINS_SIZE = 95 + 95 * 95 + 95 * 95 * 95;
//	//struct PasswordMapping* mappings;
//	//hipHostAlloc(&mappings, sizeof(struct PasswordMapping) * CHAINS_SIZE, hipHostMallocDefault);
//	//openTableFile((string("../") + "1-3#" + "ascii-32-95#" + "1").c_str(), mappings, sizeof(struct PasswordMapping), CHAINS_SIZE);
//	//for (int i = 0;i < CHAINS_SIZE;i++) {
//	//	printf("%s\n", mappings[i].hash);
//	//}
//	//getchar();
//
//	return 0;
//}

void generateTable(const uint8_t plainLength, const char* hostCharSetPath, const uint8_t plainCharSetSize)
{
	// chainSize = blockNum * threadPerBlock
	// cover = chainSize * chainLength
	const uint32_t threadPerBlock = 384;
	uint32_t blockNum = 0;

	uint32_t CHAINS_SIZE = 0;
	uint32_t chainLength = 0;
	// default plainCharSetSize == 95
	// strategy
	switch (plainLength) {
	// the collision ration is quite high, especially when the plainLength is low
	// the paper's reductant version is better when the plainLength is low (4,5,6)
	// must split the table , even when the table size is low
	case 4:
		chainLength = 350;
		CHAINS_SIZE = 384000;
		blockNum = 1000;
		break;
	case 5:
		chainLength = 3600;
		CHAINS_SIZE = 2304000;
		blockNum = 6000;
		break;
	case 6:
		chainLength = 60000;
		CHAINS_SIZE = 15360000;
		blockNum = 40000;
	}

	struct Chain* devicePointer;
	struct Chain* hostPointer;
	char* hostCharSet;
	char* deviceCharSet;
	//CUDA_CALL(hipHostAlloc(&hostPointer, CHAINS_SIZE * sizeof(struct Chain), hipHostMallocDefault | hipHostMallocMapped));
	//CUDA_CALL(hipHostAlloc(&hostCharSet, 36 * sizeof(char), hipHostMallocDefault | hipHostMallocMapped));
	CUDA_CALL(hipHostAlloc(&hostPointer, CHAINS_SIZE * sizeof(struct Chain), hipHostMallocDefault));
	CUDA_CALL(hipHostAlloc(&hostCharSet, plainCharSetSize * sizeof(char), hipHostMallocDefault));
	
	getCharSet(hostCharSet, hostCharSetPath, plainCharSetSize);
	
	generateInitialIndex(hostPointer, CHAINS_SIZE);
	
	//printf("%llu", hostPointer[0].indexS);
	
	CUDA_CALL(hipMalloc(&devicePointer, CHAINS_SIZE * sizeof(struct Chain)));
	CUDA_CALL(hipMalloc(&deviceCharSet, plainCharSetSize * sizeof(char)));
	
	CUDA_CALL(hipMemcpy(deviceCharSet, hostCharSet, plainCharSetSize * sizeof(char), hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(devicePointer, hostPointer, CHAINS_SIZE * sizeof(struct Chain), hipMemcpyHostToDevice));
	
	//CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(constantAreaPlainCharSet), hostCharSet, sizeof(char) * plainCharSetSize));
	
	ulong plainSpace = pow(plainCharSetSize, plainLength);

	hipEvent_t startEvent;
	hipEvent_t endEvent;
	float cudaElapsedTime = 0.0f;
	hipEventCreate(&startEvent);
	hipEventCreate(&endEvent);
	hipEventRecord(startEvent, 0);
	
	generateChainPaperVersion << <blockNum, threadPerBlock >> > (devicePointer, plainCharSetSize, plainLength, chainLength);
	
	hipEventRecord(endEvent, 0);
	hipEventSynchronize(endEvent);
	hipEventElapsedTime(&cudaElapsedTime, startEvent, endEvent);
	
	thrust::device_ptr<struct Chain> thrustChainP(devicePointer);
	thrust::sort(thrustChainP, thrustChainP + CHAINS_SIZE, ChainComparator());
	
	

	CUDA_CALL(hipMemcpy(hostPointer, devicePointer, CHAINS_SIZE * sizeof(struct Chain), hipMemcpyDeviceToHost));

	struct Chain* forWrite;
	CUDA_CALL(hipHostAlloc(&forWrite, sizeof(struct Chain) * CHAINS_SIZE, hipHostMallocDefault));
	uint32_t actualSize = removeDuplicate(forWrite, hostPointer, CHAINS_SIZE);

	writeToFile(fileNameBuilder("../", plainLength, hostCharSetPath, 1, actualSize, chainLength).c_str(), forWrite, sizeof(struct Chain), actualSize);
	//writeToFile("../5#ascii-32-95#1#384000#350", hostPointer, sizeof(struct Chain), CHAINS_SIZE);

	hipHostFree(hostPointer);
	hipHostFree(hostCharSet);
	hipFree(deviceCharSet);
	hipFree(devicePointer);
	//hipEventDestroy(startEvent);
	//hipEventDestroy(endEvent);
	
	hipDeviceReset();
	
	printf("%.3lf MH/S", (CHAINS_SIZE * (ulong)chainLength) / (cudaElapsedTime * 1000.0));
}

int main()
{
	generateTable(4, "../charsets/ascii-32-95.txt", 95);
	return 0;
}

//int main(int argc, char *argv[])
//{
//	const uint CHAINS_SIZE = 7680000;
//	int plainLength = 4;
//	int chainLength = 100000;
//
//	int plainCharSetSize = 95;
//
//	//hipSetDeviceFlags(hipDeviceMapHost);
//	struct Chain* devicePointer;
//	struct Chain* hostPointer;
//	char* hostCharSet;
//	char* deviceCharSet;
//	//CUDA_CALL(hipHostAlloc(&hostPointer, CHAINS_SIZE * sizeof(struct Chain), hipHostMallocDefault | hipHostMallocMapped));
//	//CUDA_CALL(hipHostAlloc(&hostCharSet, 36 * sizeof(char), hipHostMallocDefault | hipHostMallocMapped));
//	CUDA_CALL(hipHostAlloc(&hostPointer, CHAINS_SIZE * sizeof(struct Chain), hipHostMallocDefault));
//	CUDA_CALL(hipHostAlloc(&hostCharSet, plainCharSetSize * sizeof(char), hipHostMallocDefault));
//
//	getCharSet(hostCharSet, "../charsets/ascii-32-95.txt", plainCharSetSize);
//
//	generateInitialIndex(hostPointer, CHAINS_SIZE);
//
//	//printf("%llu", hostPointer[0].indexS);
//
//	CUDA_CALL(hipMalloc(&devicePointer, CHAINS_SIZE * sizeof(struct Chain)));
//	CUDA_CALL(hipMalloc(&deviceCharSet, plainCharSetSize * sizeof(char)));
//
//	CUDA_CALL(hipMemcpy(deviceCharSet, hostCharSet, plainCharSetSize * sizeof(char), hipMemcpyHostToDevice));
//	CUDA_CALL(hipMemcpy(devicePointer, hostPointer, CHAINS_SIZE * sizeof(struct Chain), hipMemcpyHostToDevice));
//
//	CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(constantAreaPlainCharSet), hostCharSet, sizeof(char) * plainCharSetSize));
//
//	/*hiprandGenerator_t randGeneratorDevice;
//	const ulong seed = 987654321;
//	const hiprandRngType_t generatorType = HIPRAND_RNG_PSEUDO_DEFAULT;
//
//	hiprandCreateGenerator(&randGeneratorDevice, generatorType);
//	hiprandSetPseudoRandomGeneratorSeed(randGeneratorDevice, seed);
//	hiprandGenerateLongLong(randGeneratorDevice, (ulong *)devicePointer, CHAINS_SIZE * 2);*/
//
//	int threadPerBlock = 384;
//	uint blockNum = CHAINS_SIZE / threadPerBlock;
//
//	hipEvent_t startEvent;
//	hipEvent_t endEvent;
//	float cudaElapsedTime = 0.0f;
//	hipEventCreate(&startEvent);
//	hipEventCreate(&endEvent);
//	hipEventRecord(startEvent, 0);
//
//	generateChainPaperVersion << <blockNum, threadPerBlock >> > (devicePointer, plainCharSetSize, plainLength, chainLength);
//
//	hipEventRecord(endEvent, 0);
//	hipEventSynchronize(endEvent);
//	hipEventElapsedTime(&cudaElapsedTime, startEvent, endEvent);
//
//	thrust::device_ptr<struct Chain> thrustChainP(devicePointer);
//	thrust::sort(thrustChainP, thrustChainP + CHAINS_SIZE, ChainComparator());
//
//	CUDA_CALL(hipMemcpy(hostPointer, devicePointer, CHAINS_SIZE * sizeof(struct Chain), hipMemcpyDeviceToHost));
//
//	// plainLength#charSet#table#tableLength#chainLength
//
//	// 1-3#ascii-32-95#1#0#chainLength
//	//writeToFile((string("../") + "1-3#" + "ascii-32-95#" + "1#" + "0#" + ).c_str(), hostPointer, sizeof(struct Chain), CHAINS_SIZE);
//	writeToFile("../t5.rt", hostPointer, sizeof(struct Chain), CHAINS_SIZE);
//
//
//	hipHostFree(hostPointer);
//	hipHostFree(hostCharSet);
//	hipFree(deviceCharSet);
//	hipFree(devicePointer);
//	//hipEventDestroy(startEvent);
//	//hipEventDestroy(endEvent);
//
//	hipDeviceReset();
//
//	printf("%.3lf MH/S", (CHAINS_SIZE * (ulong)chainLength) / (cudaElapsedTime * 1000.0));
//
//	getchar();
//
//
//
//	return 0;
//}
