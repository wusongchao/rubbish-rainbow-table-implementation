#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "round.cuh"

#include "utils.h"
#include <thread>
#include <vector>
#include <mutex>

#define INDEX_SIZE_IN_BYTES 8

#define CUDA_CALL(x) {const hipError_t a = (x);if(a!=hipSuccess){printf("\nCUDA Error:%s(err_num=%d)\n",hipGetErrorString(a),a);}}

#define EXTRACT_9 0x7fffffffffffffff
#define EXTRACT_8 0x00ffffffffffffff
#define EXTRACT_7 0x0001ffffffffffff

using std::thread;

std::mutex lock;

bool flag = false;

uint hostK[64] = {
	0x428a2f98,0x71374491,0xb5c0fbcf,0xe9b5dba5,0x3956c25b,0x59f111f1,0x923f82a4,0xab1c5ed5,
	0xd807aa98,0x12835b01,0x243185be,0x550c7dc3,0x72be5d74,0x80deb1fe,0x9bdc06a7,0xc19bf174,
	0xe49b69c1,0xefbe4786,0x0fc19dc6,0x240ca1cc,0x2de92c6f,0x4a7484aa,0x5cb0a9dc,0x76f988da,
	0x983e5152,0xa831c66d,0xb00327c8,0xbf597fc7,0xc6e00bf3,0xd5a79147,0x06ca6351,0x14292967,
	0x27b70a85,0x2e1b2138,0x4d2c6dfc,0x53380d13,0x650a7354,0x766a0abb,0x81c2c92e,0x92722c85,
	0xa2bfe8a1,0xa81a664b,0xc24b8b70,0xc76c51a3,0xd192e819,0xd6990624,0xf40e3585,0x106aa070,
	0x19a4c116,0x1e376c08,0x2748774c,0x34b0bcb5,0x391c0cb3,0x4ed8aa4a,0x5b9cca4f,0x682e6ff3,
	0x748f82ee,0x78a5636f,0x84c87814,0x8cc70208,0x90befffa,0xa4506ceb,0xbef9a3f7,0xc67178f2
};

__shared__ char plainCharSet[384][95];


//__shared__ unsigned char hashSourceShared[32];

__device__ void indexToPlain(ulong index, size_t plainCharsetSize,
	size_t plainLength, char* plain)
{
	char * plainCharSetP = plainCharSet[threadIdx.x];
	for (size_t i = 0;i < plainLength;i++) {
		plain[i] = plainCharSetP[index % plainCharsetSize];
		index /= plainCharsetSize;
	}
}

__device__ inline ulong reductFinalIndex(ulong index, uint8_t plainLength, uint8_t plainCharSize)
{
	ulong res = 0;
	uint8_t plainIndex[9];
	for (int l = plainLength - 1; l >= 0; l--) {
		plainIndex[l] = ((uint8_t)(index & 0x7f)) % plainCharSize;
		index >>= 7;
	}
	int j;
	for (j = 0; j < plainLength - 1; j++) {
		res += plainIndex[j];
		res <<= 7;
	}
	res += plainIndex[j];
	return res;
}

__device__ inline void plainToHashWithInlinePTX(ulong index, const uint8_t length, unsigned char* res, const uint8_t charSetSize) {
	unsigned int bitlen0 = 0;
	unsigned int bitlen1 = 0;
	unsigned int stateP[8];

	unsigned char data[64];

	unsigned int l;

	for (l = length - 1; l >= 1; l--) {
		data[l] = (index & 0x7f) % charSetSize;
		index >>= 7;
	}
	data[0] = (index & 0x7f) % charSetSize;
	l = length;

	stateP[0] = 0x6a09e667;
	stateP[1] = 0xbb67ae85;
	stateP[2] = 0x3c6ef372;
	stateP[3] = 0xa54ff53a;
	stateP[4] = 0x510e527f;
	stateP[5] = 0x9b05688c;
	stateP[6] = 0x1f83d9ab;
	stateP[7] = 0x5be0cd19;


	data[l++] = 0x80;
	while (l < 56)
		data[l++] = 0x00;


	//// Append to the padding the total message's length in bits and transform. 
	DBL_INT_ADD(bitlen0, bitlen1, length * 8);

	data[63] = bitlen0;
	data[62] = bitlen0 >> 8;
	data[61] = bitlen0 >> 16;
	data[60] = bitlen0 >> 24;
	data[59] = bitlen1;
	data[58] = bitlen1 >> 8;
	data[57] = bitlen1 >> 16;
	data[56] = bitlen1 >> 24;

	uint32_t schedule[16];

	uint32_t a = stateP[0];
	uint32_t b = stateP[1];
	uint32_t c = stateP[2];
	uint32_t d = stateP[3];
	uint32_t e = stateP[4];
	uint32_t f = stateP[5];
	uint32_t g = stateP[6];
	uint32_t h = stateP[7];

	ROUNDa(0, a, b, c, d, e, f, g, h, 0x428A2F98)
		ROUNDa(1, h, a, b, c, d, e, f, g, 0x71374491)
		ROUNDa(2, g, h, a, b, c, d, e, f, -0x4A3F0431)
		ROUNDa(3, f, g, h, a, b, c, d, e, -0x164A245B)
		ROUNDa(4, e, f, g, h, a, b, c, d, 0x3956C25B)
		ROUNDa(5, d, e, f, g, h, a, b, c, 0x59F111F1)
		ROUNDa(6, c, d, e, f, g, h, a, b, -0x6DC07D5C)
		ROUNDa(7, b, c, d, e, f, g, h, a, -0x54E3A12B)
		ROUNDa(8, a, b, c, d, e, f, g, h, -0x27F85568)
		ROUNDa(9, h, a, b, c, d, e, f, g, 0x12835B01)
		ROUNDa(10, g, h, a, b, c, d, e, f, 0x243185BE)
		ROUNDa(11, f, g, h, a, b, c, d, e, 0x550C7DC3)
		ROUNDa(12, e, f, g, h, a, b, c, d, 0x72BE5D74)
		ROUNDa(13, d, e, f, g, h, a, b, c, -0x7F214E02)
		ROUNDa(14, c, d, e, f, g, h, a, b, -0x6423F959)
		ROUNDa(15, b, c, d, e, f, g, h, a, -0x3E640E8C)
		ROUND16(16, a, b, c, d, e, f, g, h, -0x1B64963F)
		ROUND17(17, h, a, b, c, d, e, f, g, -0x1041B87A)
		ROUND18(18, g, h, a, b, c, d, e, f, 0x0FC19DC6)
		ROUND19(19, f, g, h, a, b, c, d, e, 0x240CA1CC)
		ROUND20(20, e, f, g, h, a, b, c, d, 0x2DE92C6F)
		ROUND21(21, d, e, f, g, h, a, b, c, 0x4A7484AA)
		ROUND22(22, c, d, e, f, g, h, a, b, 0x5CB0A9DC)
		ROUND23(23, b, c, d, e, f, g, h, a, 0x76F988DA)
		ROUND24(24, a, b, c, d, e, f, g, h, -0x67C1AEAE)
		ROUND25(25, h, a, b, c, d, e, f, g, -0x57CE3993)
		ROUND26(26, g, h, a, b, c, d, e, f, -0x4FFCD838)
		ROUND27(27, f, g, h, a, b, c, d, e, -0x40A68039)
		ROUND28(28, e, f, g, h, a, b, c, d, -0x391FF40D)
		ROUND29(29, d, e, f, g, h, a, b, c, -0x2A586EB9)
		ROUND30(30, c, d, e, f, g, h, a, b, 0x06CA6351)
		ROUND31(31, b, c, d, e, f, g, h, a, 0x14292967)
		ROUND16(32, a, b, c, d, e, f, g, h, 0x27B70A85)
		ROUND17(33, h, a, b, c, d, e, f, g, 0x2E1B2138)
		ROUND18(34, g, h, a, b, c, d, e, f, 0x4D2C6DFC)
		ROUND19(35, f, g, h, a, b, c, d, e, 0x53380D13)
		ROUND20(36, e, f, g, h, a, b, c, d, 0x650A7354)
		ROUND21(37, d, e, f, g, h, a, b, c, 0x766A0ABB)
		ROUND22(38, c, d, e, f, g, h, a, b, -0x7E3D36D2)
		ROUND23(39, b, c, d, e, f, g, h, a, -0x6D8DD37B)
		ROUND24(40, a, b, c, d, e, f, g, h, -0x5D40175F)
		ROUND25(41, h, a, b, c, d, e, f, g, -0x57E599B5)
		ROUND26(42, g, h, a, b, c, d, e, f, -0x3DB47490)
		ROUND27(43, f, g, h, a, b, c, d, e, -0x3893AE5D)
		ROUND28(44, e, f, g, h, a, b, c, d, -0x2E6D17E7)
		ROUND29(45, d, e, f, g, h, a, b, c, -0x2966F9DC)
		ROUND30(46, c, d, e, f, g, h, a, b, -0x0BF1CA7B)
		ROUND31(47, b, c, d, e, f, g, h, a, 0x106AA070)
		ROUND16(48, a, b, c, d, e, f, g, h, 0x19A4C116)
		ROUND17(49, h, a, b, c, d, e, f, g, 0x1E376C08)
		ROUND18(50, g, h, a, b, c, d, e, f, 0x2748774C)
		ROUND19(51, f, g, h, a, b, c, d, e, 0x34B0BCB5)
		ROUND20(52, e, f, g, h, a, b, c, d, 0x391C0CB3)
		ROUND21(53, d, e, f, g, h, a, b, c, 0x4ED8AA4A)
		ROUND22(54, c, d, e, f, g, h, a, b, 0x5B9CCA4F)
		ROUND23(55, b, c, d, e, f, g, h, a, 0x682E6FF3)
		ROUND24(56, a, b, c, d, e, f, g, h, 0x748F82EE)
		ROUND25(57, h, a, b, c, d, e, f, g, 0x78A5636F)
		ROUND26(58, g, h, a, b, c, d, e, f, -0x7B3787EC)
		ROUND27(59, f, g, h, a, b, c, d, e, -0x7338FDF8)
		ROUND28(60, e, f, g, h, a, b, c, d, -0x6F410006)
		ROUND29(61, d, e, f, g, h, a, b, c, -0x5BAF9315)
		ROUND30(62, c, d, e, f, g, h, a, b, -0x41065C09)
		ROUND31(63, b, c, d, e, f, g, h, a, -0x398E870E)

		stateP[0] += a;
	stateP[1] += b;
	stateP[2] += c;
	stateP[3] += d;
	stateP[4] += e;
	stateP[5] += f;
	stateP[6] += g;
	stateP[7] += h;

	// Since this implementation uses little endian byte ordering and SHA uses big endian,
	// reverse all the bytes when copying the final state to the output hash. 

	for (int i = 0; i < 4; ++i) {
		l = i << 3;
		*(res) = (stateP[0] >> (24 - l)) & 0x000000ff;
		*(res + 4) = (stateP[1] >> (24 - l)) & 0x000000ff;
		*(res + 8) = (stateP[2] >> (24 - l)) & 0x000000ff;
		*(res + 12) = (stateP[3] >> (24 - l)) & 0x000000ff;
		*(res + 16) = (stateP[4] >> (24 - l)) & 0x000000ff;
		*(res + 20) = (stateP[5] >> (24 - l)) & 0x000000ff;
		*(res + 24) = (stateP[6] >> (24 - l)) & 0x000000ff;
		*(res + 28) = (stateP[7] >> (24 - l)) & 0x000000ff;
		++res;
	}
}

__device__ inline void plainToHashWithInlinePTX(char* plain, const unsigned int length, unsigned char* res) {
	unsigned int bitlen0 = 0;
	unsigned int bitlen1 = 0;
	unsigned int stateP[8];

	//unsigned int* stateP = state[threadIdx.x];

	unsigned char data[64];

	unsigned int l;

	for (l = 0; l < length; ++l) {
		data[l] = plain[l];
	}


	stateP[0] = 0x6a09e667;
	stateP[1] = 0xbb67ae85;
	stateP[2] = 0x3c6ef372;
	stateP[3] = 0xa54ff53a;
	stateP[4] = 0x510e527f;
	stateP[5] = 0x9b05688c;
	stateP[6] = 0x1f83d9ab;
	stateP[7] = 0x5be0cd19;


	data[l++] = 0x80;
	while (l < 56)
		data[l++] = 0x00;


	//// Append to the padding the total message's length in bits and transform. 
	DBL_INT_ADD(bitlen0, bitlen1, length * 8);

	data[63] = bitlen0;
	data[62] = bitlen0 >> 8;
	data[61] = bitlen0 >> 16;
	data[60] = bitlen0 >> 24;
	data[59] = bitlen1;
	data[58] = bitlen1 >> 8;
	data[57] = bitlen1 >> 16;
	data[56] = bitlen1 >> 24;

	uint32_t schedule[16];

	uint32_t a = stateP[0];
	uint32_t b = stateP[1];
	uint32_t c = stateP[2];
	uint32_t d = stateP[3];
	uint32_t e = stateP[4];
	uint32_t f = stateP[5];
	uint32_t g = stateP[6];
	uint32_t h = stateP[7];

	ROUNDa(0, a, b, c, d, e, f, g, h, 0x428A2F98)
		ROUNDa(1, h, a, b, c, d, e, f, g, 0x71374491)
		ROUNDa(2, g, h, a, b, c, d, e, f, -0x4A3F0431)
		ROUNDa(3, f, g, h, a, b, c, d, e, -0x164A245B)
		ROUNDa(4, e, f, g, h, a, b, c, d, 0x3956C25B)
		ROUNDa(5, d, e, f, g, h, a, b, c, 0x59F111F1)
		ROUNDa(6, c, d, e, f, g, h, a, b, -0x6DC07D5C)
		ROUNDa(7, b, c, d, e, f, g, h, a, -0x54E3A12B)
		ROUNDa(8, a, b, c, d, e, f, g, h, -0x27F85568)
		ROUNDa(9, h, a, b, c, d, e, f, g, 0x12835B01)
		ROUNDa(10, g, h, a, b, c, d, e, f, 0x243185BE)
		ROUNDa(11, f, g, h, a, b, c, d, e, 0x550C7DC3)
		ROUNDa(12, e, f, g, h, a, b, c, d, 0x72BE5D74)
		ROUNDa(13, d, e, f, g, h, a, b, c, -0x7F214E02)
		ROUNDa(14, c, d, e, f, g, h, a, b, -0x6423F959)
		ROUNDa(15, b, c, d, e, f, g, h, a, -0x3E640E8C)
		ROUND16(16, a, b, c, d, e, f, g, h, -0x1B64963F)
		ROUND17(17, h, a, b, c, d, e, f, g, -0x1041B87A)
		ROUND18(18, g, h, a, b, c, d, e, f, 0x0FC19DC6)
		ROUND19(19, f, g, h, a, b, c, d, e, 0x240CA1CC)
		ROUND20(20, e, f, g, h, a, b, c, d, 0x2DE92C6F)
		ROUND21(21, d, e, f, g, h, a, b, c, 0x4A7484AA)
		ROUND22(22, c, d, e, f, g, h, a, b, 0x5CB0A9DC)
		ROUND23(23, b, c, d, e, f, g, h, a, 0x76F988DA)
		ROUND24(24, a, b, c, d, e, f, g, h, -0x67C1AEAE)
		ROUND25(25, h, a, b, c, d, e, f, g, -0x57CE3993)
		ROUND26(26, g, h, a, b, c, d, e, f, -0x4FFCD838)
		ROUND27(27, f, g, h, a, b, c, d, e, -0x40A68039)
		ROUND28(28, e, f, g, h, a, b, c, d, -0x391FF40D)
		ROUND29(29, d, e, f, g, h, a, b, c, -0x2A586EB9)
		ROUND30(30, c, d, e, f, g, h, a, b, 0x06CA6351)
		ROUND31(31, b, c, d, e, f, g, h, a, 0x14292967)
		ROUND16(32, a, b, c, d, e, f, g, h, 0x27B70A85)
		ROUND17(33, h, a, b, c, d, e, f, g, 0x2E1B2138)
		ROUND18(34, g, h, a, b, c, d, e, f, 0x4D2C6DFC)
		ROUND19(35, f, g, h, a, b, c, d, e, 0x53380D13)
		ROUND20(36, e, f, g, h, a, b, c, d, 0x650A7354)
		ROUND21(37, d, e, f, g, h, a, b, c, 0x766A0ABB)
		ROUND22(38, c, d, e, f, g, h, a, b, -0x7E3D36D2)
		ROUND23(39, b, c, d, e, f, g, h, a, -0x6D8DD37B)
		ROUND24(40, a, b, c, d, e, f, g, h, -0x5D40175F)
		ROUND25(41, h, a, b, c, d, e, f, g, -0x57E599B5)
		ROUND26(42, g, h, a, b, c, d, e, f, -0x3DB47490)
		ROUND27(43, f, g, h, a, b, c, d, e, -0x3893AE5D)
		ROUND28(44, e, f, g, h, a, b, c, d, -0x2E6D17E7)
		ROUND29(45, d, e, f, g, h, a, b, c, -0x2966F9DC)
		ROUND30(46, c, d, e, f, g, h, a, b, -0x0BF1CA7B)
		ROUND31(47, b, c, d, e, f, g, h, a, 0x106AA070)
		ROUND16(48, a, b, c, d, e, f, g, h, 0x19A4C116)
		ROUND17(49, h, a, b, c, d, e, f, g, 0x1E376C08)
		ROUND18(50, g, h, a, b, c, d, e, f, 0x2748774C)
		ROUND19(51, f, g, h, a, b, c, d, e, 0x34B0BCB5)
		ROUND20(52, e, f, g, h, a, b, c, d, 0x391C0CB3)
		ROUND21(53, d, e, f, g, h, a, b, c, 0x4ED8AA4A)
		ROUND22(54, c, d, e, f, g, h, a, b, 0x5B9CCA4F)
		ROUND23(55, b, c, d, e, f, g, h, a, 0x682E6FF3)
		ROUND24(56, a, b, c, d, e, f, g, h, 0x748F82EE)
		ROUND25(57, h, a, b, c, d, e, f, g, 0x78A5636F)
		ROUND26(58, g, h, a, b, c, d, e, f, -0x7B3787EC)
		ROUND27(59, f, g, h, a, b, c, d, e, -0x7338FDF8)
		ROUND28(60, e, f, g, h, a, b, c, d, -0x6F410006)
		ROUND29(61, d, e, f, g, h, a, b, c, -0x5BAF9315)
		ROUND30(62, c, d, e, f, g, h, a, b, -0x41065C09)
		ROUND31(63, b, c, d, e, f, g, h, a, -0x398E870E)

		stateP[0] += a;
	stateP[1] += b;
	stateP[2] += c;
	stateP[3] += d;
	stateP[4] += e;
	stateP[5] += f;
	stateP[6] += g;
	stateP[7] += h;

	// Since this implementation uses little endian byte ordering and SHA uses big endian,
	// reverse all the bytes when copying the final state to the output hash. 

	for (int i = 0; i < 4; ++i) {
		l = i << 3;
		*(res) = (stateP[0] >> (24 - l)) & 0x000000ff;
		*(res + 4) = (stateP[1] >> (24 - l)) & 0x000000ff;
		*(res + 8) = (stateP[2] >> (24 - l)) & 0x000000ff;
		*(res + 12) = (stateP[3] >> (24 - l)) & 0x000000ff;
		*(res + 16) = (stateP[4] >> (24 - l)) & 0x000000ff;
		*(res + 20) = (stateP[5] >> (24 - l)) & 0x000000ff;
		*(res + 24) = (stateP[6] >> (24 - l)) & 0x000000ff;
		*(res + 28) = (stateP[7] >> (24 - l)) & 0x000000ff;
		++res;
	}

}

__device__ inline void plainToHash(ulong index, const uint8_t length, unsigned char* res, const uint8_t charSetSize)
{
	unsigned int bitlen0 = 0;
	unsigned int bitlen1 = 0;
	unsigned int stateP[8];

	unsigned char data[64];

	unsigned int l;

	for (l = length - 1; l >= 1; l--) {
		data[l] = (index & 0x7f) % charSetSize;
		index >>= 7;
	}
	data[0] = (index & 0x7f) % charSetSize;
	l = length;

	stateP[0] = 0x6a09e667;
	stateP[1] = 0xbb67ae85;
	stateP[2] = 0x3c6ef372;
	stateP[3] = 0xa54ff53a;
	stateP[4] = 0x510e527f;
	stateP[5] = 0x9b05688c;
	stateP[6] = 0x1f83d9ab;
	stateP[7] = 0x5be0cd19;


	// Pad whatever data is left in the buffer. 
	data[l++] = 0x80;
	while (l < 56)
		data[l++] = 0x00;


	// Append to the padding the total message's length in bits and transform. 
	DBL_INT_ADD(bitlen0, bitlen1, length * 8);
	data[63] = bitlen0;
	data[62] = bitlen0 >> 8;
	data[61] = bitlen0 >> 16;
	data[60] = bitlen0 >> 24;
	data[59] = bitlen1;
	data[58] = bitlen1 >> 8;
	data[57] = bitlen1 >> 16;
	data[56] = bitlen1 >> 24;


	uint32_t schedule[64];
	LOADSCHEDULE(0)
		LOADSCHEDULE(1)
		LOADSCHEDULE(2)
		LOADSCHEDULE(3)
		LOADSCHEDULE(4)
		LOADSCHEDULE(5)
		LOADSCHEDULE(6)
		LOADSCHEDULE(7)
		LOADSCHEDULE(8)
		LOADSCHEDULE(9)
		LOADSCHEDULE(10)
		LOADSCHEDULE(11)
		LOADSCHEDULE(12)
		LOADSCHEDULE(13)
		LOADSCHEDULE(14)
		LOADSCHEDULE(15)
		SCHEDULE(16)
		SCHEDULE(17)
		SCHEDULE(18)
		SCHEDULE(19)
		SCHEDULE(20)
		SCHEDULE(21)
		SCHEDULE(22)
		SCHEDULE(23)
		SCHEDULE(24)
		SCHEDULE(25)
		SCHEDULE(26)
		SCHEDULE(27)
		SCHEDULE(28)
		SCHEDULE(29)
		SCHEDULE(30)
		SCHEDULE(31)
		SCHEDULE(32)
		SCHEDULE(33)
		SCHEDULE(34)
		SCHEDULE(35)
		SCHEDULE(36)
		SCHEDULE(37)
		SCHEDULE(38)
		SCHEDULE(39)
		SCHEDULE(40)
		SCHEDULE(41)
		SCHEDULE(42)
		SCHEDULE(43)
		SCHEDULE(44)
		SCHEDULE(45)
		SCHEDULE(46)
		SCHEDULE(47)
		SCHEDULE(48)
		SCHEDULE(49)
		SCHEDULE(50)
		SCHEDULE(51)
		SCHEDULE(52)
		SCHEDULE(53)
		SCHEDULE(54)
		SCHEDULE(55)
		SCHEDULE(56)
		SCHEDULE(57)
		SCHEDULE(58)
		SCHEDULE(59)
		SCHEDULE(60)
		SCHEDULE(61)
		SCHEDULE(62)
		SCHEDULE(63)

		uint32_t a = stateP[0];
	uint32_t b = stateP[1];
	uint32_t c = stateP[2];
	uint32_t d = stateP[3];
	uint32_t e = stateP[4];
	uint32_t f = stateP[5];
	uint32_t g = stateP[6];
	uint32_t h = stateP[7];
	ROUND(a, b, c, d, e, f, g, h, 0, 0x428A2F98)
		ROUND(h, a, b, c, d, e, f, g, 1, 0x71374491)
		ROUND(g, h, a, b, c, d, e, f, 2, 0xB5C0FBCF)
		ROUND(f, g, h, a, b, c, d, e, 3, 0xE9B5DBA5)
		ROUND(e, f, g, h, a, b, c, d, 4, 0x3956C25B)
		ROUND(d, e, f, g, h, a, b, c, 5, 0x59F111F1)
		ROUND(c, d, e, f, g, h, a, b, 6, 0x923F82A4)
		ROUND(b, c, d, e, f, g, h, a, 7, 0xAB1C5ED5)
		ROUND(a, b, c, d, e, f, g, h, 8, 0xD807AA98)
		ROUND(h, a, b, c, d, e, f, g, 9, 0x12835B01)
		ROUND(g, h, a, b, c, d, e, f, 10, 0x243185BE)
		ROUND(f, g, h, a, b, c, d, e, 11, 0x550C7DC3)
		ROUND(e, f, g, h, a, b, c, d, 12, 0x72BE5D74)
		ROUND(d, e, f, g, h, a, b, c, 13, 0x80DEB1FE)
		ROUND(c, d, e, f, g, h, a, b, 14, 0x9BDC06A7)
		ROUND(b, c, d, e, f, g, h, a, 15, 0xC19BF174)
		ROUND(a, b, c, d, e, f, g, h, 16, 0xE49B69C1)
		ROUND(h, a, b, c, d, e, f, g, 17, 0xEFBE4786)
		ROUND(g, h, a, b, c, d, e, f, 18, 0x0FC19DC6)
		ROUND(f, g, h, a, b, c, d, e, 19, 0x240CA1CC)
		ROUND(e, f, g, h, a, b, c, d, 20, 0x2DE92C6F)
		ROUND(d, e, f, g, h, a, b, c, 21, 0x4A7484AA)
		ROUND(c, d, e, f, g, h, a, b, 22, 0x5CB0A9DC)
		ROUND(b, c, d, e, f, g, h, a, 23, 0x76F988DA)
		ROUND(a, b, c, d, e, f, g, h, 24, 0x983E5152)
		ROUND(h, a, b, c, d, e, f, g, 25, 0xA831C66D)
		ROUND(g, h, a, b, c, d, e, f, 26, 0xB00327C8)
		ROUND(f, g, h, a, b, c, d, e, 27, 0xBF597FC7)
		ROUND(e, f, g, h, a, b, c, d, 28, 0xC6E00BF3)
		ROUND(d, e, f, g, h, a, b, c, 29, 0xD5A79147)
		ROUND(c, d, e, f, g, h, a, b, 30, 0x06CA6351)
		ROUND(b, c, d, e, f, g, h, a, 31, 0x14292967)
		ROUND(a, b, c, d, e, f, g, h, 32, 0x27B70A85)
		ROUND(h, a, b, c, d, e, f, g, 33, 0x2E1B2138)
		ROUND(g, h, a, b, c, d, e, f, 34, 0x4D2C6DFC)
		ROUND(f, g, h, a, b, c, d, e, 35, 0x53380D13)
		ROUND(e, f, g, h, a, b, c, d, 36, 0x650A7354)
		ROUND(d, e, f, g, h, a, b, c, 37, 0x766A0ABB)
		ROUND(c, d, e, f, g, h, a, b, 38, 0x81C2C92E)
		ROUND(b, c, d, e, f, g, h, a, 39, 0x92722C85)
		ROUND(a, b, c, d, e, f, g, h, 40, 0xA2BFE8A1)
		ROUND(h, a, b, c, d, e, f, g, 41, 0xA81A664B)
		ROUND(g, h, a, b, c, d, e, f, 42, 0xC24B8B70)
		ROUND(f, g, h, a, b, c, d, e, 43, 0xC76C51A3)
		ROUND(e, f, g, h, a, b, c, d, 44, 0xD192E819)
		ROUND(d, e, f, g, h, a, b, c, 45, 0xD6990624)
		ROUND(c, d, e, f, g, h, a, b, 46, 0xF40E3585)
		ROUND(b, c, d, e, f, g, h, a, 47, 0x106AA070)
		ROUND(a, b, c, d, e, f, g, h, 48, 0x19A4C116)
		ROUND(h, a, b, c, d, e, f, g, 49, 0x1E376C08)
		ROUND(g, h, a, b, c, d, e, f, 50, 0x2748774C)
		ROUND(f, g, h, a, b, c, d, e, 51, 0x34B0BCB5)
		ROUND(e, f, g, h, a, b, c, d, 52, 0x391C0CB3)
		ROUND(d, e, f, g, h, a, b, c, 53, 0x4ED8AA4A)
		ROUND(c, d, e, f, g, h, a, b, 54, 0x5B9CCA4F)
		ROUND(b, c, d, e, f, g, h, a, 55, 0x682E6FF3)
		ROUND(a, b, c, d, e, f, g, h, 56, 0x748F82EE)
		ROUND(h, a, b, c, d, e, f, g, 57, 0x78A5636F)
		ROUND(g, h, a, b, c, d, e, f, 58, 0x84C87814)
		ROUND(f, g, h, a, b, c, d, e, 59, 0x8CC70208)
		ROUND(e, f, g, h, a, b, c, d, 60, 0x90BEFFFA)
		ROUND(d, e, f, g, h, a, b, c, 61, 0xA4506CEB)
		ROUND(c, d, e, f, g, h, a, b, 62, 0xBEF9A3F7)
		ROUND(b, c, d, e, f, g, h, a, 63, 0xC67178F2)
		stateP[0] += a;
	stateP[1] += b;
	stateP[2] += c;
	stateP[3] += d;
	stateP[4] += e;
	stateP[5] += f;
	stateP[6] += g;
	stateP[7] += h;

	// Since this implementation uses little endian byte ordering and SHA uses big endian,
	// reverse all the bytes when copying the final state to the output hash. 

	for (int i = 0; i < 4; ++i) {
		l = i << 3;
		*(res) = (stateP[0] >> (24 - l)) & 0x000000ff;
		*(res + 4) = (stateP[1] >> (24 - l)) & 0x000000ff;
		*(res + 8) = (stateP[2] >> (24 - l)) & 0x000000ff;
		*(res + 12) = (stateP[3] >> (24 - l)) & 0x000000ff;
		*(res + 16) = (stateP[4] >> (24 - l)) & 0x000000ff;
		*(res + 20) = (stateP[5] >> (24 - l)) & 0x000000ff;
		*(res + 24) = (stateP[6] >> (24 - l)) & 0x000000ff;
		*(res + 28) = (stateP[7] >> (24 - l)) & 0x000000ff;
		++res;
	}
}

__device__ void indexToPlain(ulong index, char* plain, size_t plainLength, const char* charSet, size_t charSetSize)
{
	for (int i = plainLength - 1; i >= 0; i--) {
		plain[i] = charSet[(index & 0x7f) % charSetSize];
		index >>= 7;
	}
}

__device__ inline ulong hashToIndexPaperVersion(unsigned char* hash, int pos, const unsigned int plainCharSetSize)
{
	unsigned int* hashP = (unsigned int*)hash;
	unsigned int p0 = *(hashP + 4) ^ pos;
	unsigned int p2 = *(hashP + 5) ^ (pos >> 12);
	unsigned int p4 = *(hashP + 6) ^ (pos >> 24);
	unsigned int p6 = *(hashP + 7);
	char* plainCharSetP = plainCharSet[threadIdx.x];
	unsigned __int16 b0 = plainCharSetP[p0 % plainCharSetSize] << 8 | plainCharSetP[(p0 >> 16) % plainCharSetSize];
	unsigned __int16 b1 = plainCharSetP[p2 % plainCharSetSize] << 8 | plainCharSetP[(p2 >> 16) % plainCharSetSize];
	unsigned __int16 b2 = plainCharSetP[p4 % plainCharSetSize] << 8 | plainCharSetP[(p4 >> 16) % plainCharSetSize];
	unsigned __int16 b3 = plainCharSetP[p6 % plainCharSetSize] << 8 | plainCharSetP[(p6 >> 16) % plainCharSetSize];
	ulong index = 0;
	index += b0;
	index <<= 16;
	index += b1;
	index <<= 16;
	index += b2;
	index <<= 16;
	index += b3;
	return index;
}

__device__ inline ulong hashToIndexWithoutCharSet(unsigned char* hash, int pos, const uint8_t plainCharSetSize)
{
	unsigned int* hashP = (unsigned int*)hash;
	unsigned int p0 = *(hashP + 4) ^ pos;
	unsigned int p2 = *(hashP + 5) ^ (pos >> 12);
	unsigned int p4 = *(hashP + 6) ^ (pos >> 24);
	unsigned int p6 = *(hashP + 7);

	unsigned __int16 b0 = ((p0 % plainCharSetSize) << 8) | ((p0 >> 16) % plainCharSetSize);
	unsigned __int16 b1 = ((p2 % plainCharSetSize) << 8) | ((p2 >> 16) % plainCharSetSize);
	unsigned __int16 b2 = ((p4 % plainCharSetSize) << 8) | ((p4 >> 16) % plainCharSetSize);
	unsigned __int16 b3 = ((p6 % plainCharSetSize) << 8) | ((p6 >> 16) % plainCharSetSize);

	ulong index = 0;
	index += b0;
	index <<= 16;
	index += b1;
	index <<= 16;
	index += b2;
	index <<= 16;
	index += b3;
	return index;
}

__device__ inline ulong hashToIndex(unsigned char* hash, int pos)
{
	ulong* hashP = (ulong*)hash;
	return (ulong)(((*(hashP) ^ *(hashP + 1) ^ *(hashP + 2) ^ *(hashP + 3)) + pos));
}

__device__ inline void initSHA256ConstantAndCharSet(const char* srcCharSet, const unsigned int charSetSize)
{
	char* plainCharSetP = plainCharSet[threadIdx.x];
	for (int i = 0;i < charSetSize;i++) {
		plainCharSetP[i] = srcCharSet[i];
	}
}

__global__ void calIndexFromSpecificPos(struct DecryptedInfo* decryptedInfo, unsigned char* hashSource, const unsigned int plainCharSetSize, const uint8_t plainLength, const unsigned int chainLength)
{
	uint offset = (blockIdx.x * blockDim.x) + threadIdx.x;

	//initSHA256ConstantAndCharSet(srcCharSet, plainCharSetSize);

	if (offset < chainLength) {
		unsigned char hash[32];
		for (int i = 0;i < 32;i++) {
			hash[i] = hashSource[i];
		}

		ulong indexS = hashToIndex(hash, 0);
		(decryptedInfo + offset)->pos = offset;
		
		for (int j = offset + 1;j < chainLength;j++) {
		//	plainToHashWithInlinePTX(indexS, plainLength, hash, plainCharSetSize);
		//	//plainToHashWithInlinePTX((char*)&indexS, INDEX_SIZE_IN_BYTES, hash);
			plainToHash(indexS, plainLength, hash, plainCharSetSize);
			indexS = hashToIndex(hash, j);
		//	//indexS = hashToIndexPaperVersion(hash, i, plainCharSetSize);
		}
		(decryptedInfo + offset)->index = reductFinalIndex(indexS,plainLength,plainCharSetSize);
	}
}

void threadTest(int i)
{
	printf("%d", i);
}

int main()
{
	const uint CHAINS_SIZE = 7680000;

	unsigned int chainLength = 100000;

	unsigned char givenHash[32];
	uint8_t plainLength = 6;
	//plainToHashCPU("621121", plainLength, givenHash);
	char plain[9];

	struct Chain* hostChain;
	char* hostCharSet;
	struct DecryptedInfo* hostDecryptedInfo;
	struct DecryptedInfo* deviceDecryptedInfo;
	char* deviceCharSet;
	unsigned char* deviceGivenHash;

	unsigned int plainCharSetSize = 95;
	//ulong indexS = hashToIndexCPU(givenHash, chainLength, plainSpaceTotal);

	uint threadPerBlock = 384;
	uint blockNum = chainLength / threadPerBlock + 1;

	CUDA_CALL(hipHostAlloc(&hostChain, CHAINS_SIZE * sizeof(struct Chain), hipHostMallocDefault));
	CUDA_CALL(hipHostAlloc(&hostCharSet, plainCharSetSize * sizeof(char), hipHostMallocDefault));
	CUDA_CALL(hipHostAlloc(&hostDecryptedInfo, sizeof(struct DecryptedInfo) * (chainLength), hipHostMallocDefault));

	getCharSet(hostCharSet, "../charsets/ascii-32-95.txt", plainCharSetSize);
	
	std::map<const char, size_t> charMap;

	for (int i = 0;i < plainCharSetSize;i++) {
		charMap.insert(std::make_pair<>(hostCharSet[i],i));
	}
	ulong plainIndex = plainToIndexCPU("62qq41", plainLength, hostCharSet, plainCharSetSize, &charMap);
	plainToHashCPU(plainIndex, plainLength, givenHash, plainCharSetSize);

	CUDA_CALL(hipMalloc(&deviceCharSet, plainCharSetSize * sizeof(char)));
	CUDA_CALL(hipMalloc(&deviceGivenHash, 32 * sizeof(unsigned char)));

	CUDA_CALL(hipMemcpy(deviceCharSet, hostCharSet, plainCharSetSize * sizeof(char), hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(deviceGivenHash, givenHash, 32 * sizeof(unsigned char), hipMemcpyHostToDevice));

	openTableFile("../t5.rt", hostChain, sizeof(struct Chain), CHAINS_SIZE);

	CUDA_CALL(hipMalloc(&deviceDecryptedInfo, sizeof(struct DecryptedInfo) * (chainLength)));

	hipEvent_t startEvent;
	hipEvent_t endEvent;
	float cudaElapsedTime = 0.0f;
	hipEventCreate(&startEvent);
	hipEventCreate(&endEvent);
	hipEventRecord(startEvent, 0);

	calIndexFromSpecificPos << <blockNum, threadPerBlock >> >(deviceDecryptedInfo, deviceGivenHash, plainCharSetSize, plainLength, chainLength);

	hipEventRecord(endEvent, 0);
	hipEventSynchronize(endEvent);
	hipEventElapsedTime(&cudaElapsedTime, startEvent, endEvent);

	CUDA_CALL(hipMemcpy(hostDecryptedInfo, deviceDecryptedInfo, sizeof(struct DecryptedInfo) * (chainLength), hipMemcpyDeviceToHost));

	printf("%f\n", cudaElapsedTime);

	//char res[9];
	//printf("%llx\n", hostDecryptedInfo[chainLength-1].pos);

	uint cpuThreadNum = 4;
	std::mutex;
	uint beg = 0;
	uint gap = chainLength / cpuThreadNum;

	char resultStore[4][9] = {' '};

	for (int i = 0;i < cpuThreadNum;i++) {
		std::thread t(searchAndRebuildPerThread, beg, beg + gap, hostDecryptedInfo, hostChain, CHAINS_SIZE, givenHash, plainCharSetSize, hostCharSet, plainLength,resultStore[i]);
		beg += gap;
		t.join();
	}

	printf("----------\n");

	for (int i = 0;i < cpuThreadNum;i++) {
		for (int j = 0;j < plainLength;j++) {
			putchar(resultStore[i][j]);
		}
		putchar('\n');
	}
	// thread0 : hostDecryptedInfo 0~1/4
	//for (int i = 0;i < chainLength;i++) {
	//	int pos = searchThroughChains(hostChain, CHAINS_SIZE, hostDecryptedInfo[i].index);

	//	if (pos != -1) {
	//		if (rebuildAndCompare(res, givenHash, hostChain[pos].indexS, pos, plainCharSetSize, hostCharSet, plainLength)) {
	//			putchar(res[0]);
	//			putchar(res[1]);
	//			putchar(res[2]);
	//			putchar(res[3]);
	//			putchar(res[4]);
	//			putchar(res[5]);
	//			putchar(res[6]);
	//			putchar(res[7]);
	//			putchar(res[8]);
	//			break;
	//		}
	//	}
	//}



	//hipFree(deviceDecryptedInfo);
	//hipHostFree(hostChain);
	//hipHostFree(hostCharSet);
	//hipHostFree(hostDecryptedInfo);

	/*std::vector<thread> searchThreads;
	for (int i = 0;i < 4;i++) {
	thread t(threadTest, i);
	searchThreads.push_back(std::move(t));
	}
	for (int i = 0;i < 4;i++) {
	searchThreads[i].join();
	}*/

	return 0;
}