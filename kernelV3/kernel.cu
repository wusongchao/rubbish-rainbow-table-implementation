#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "math_functions.h"
#include "round.cuh"

#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include "utils.h"
#include <thrust\device_ptr.h>
#include <thrust\sort.h>
#include <map>
#include <string>

//#pragma comment(lib, "hiprand.lib")

using std::map;

#define INDEX_SIZE_IN_BYTES 8
#define EXTRACT_9 0x7fffffffffffffff
#define EXTRACT_8 0x00ffffffffffffff
#define EXTRACT_7 0x0001ffffffffffff
#define EXTRACT_6 0x000003ffffffffff

#define ROTR32(x, n)  (((0U + (x)) << (32 - (n))) | ((x) >> (n)))  // Assumes that x is uint32_t and 0 < n < 32

#define LOADSCHEDULE(i)  \
		schedule[i] = (uint32_t)data[i * 4 + 0] << 24  \
		            | (uint32_t)data[i * 4 + 1] << 16  \
		            | (uint32_t)data[i * 4 + 2] <<  8  \
		            | (uint32_t)data[i * 4 + 3] <<  0;

#define SCHEDULE(i)  \
		schedule[i] = 0U + schedule[i - 16] + schedule[i - 7]  \
			+ (ROTR32(schedule[i - 15], 7) ^ ROTR32(schedule[i - 15], 18) ^ (schedule[i - 15] >> 3))  \
			+ (ROTR32(schedule[i - 2], 17) ^ ROTR32(schedule[i - 2], 19) ^ (schedule[i - 2] >> 10));

//#define SCHEDULE(i) \
//	asm("{\n\t" \
//		".reg .u32 t1;\n\t" \
//		".reg .u32 t2;\n\t" \
//		".reg .u32 t3;\n\t" \
//		".reg .u32 s1;\n\t" \
//		".reg .u32 s2;\n\t" \
//		".reg .u32 s3;\n\t" \
//		".reg .u32 s4;\n\t" \
//		"mov.u32 s1, %1;\n\t" \
//		"mov.u32 s2, %2;\n\t" \
//		"mov.u32 s3, %3;\n\t" \
//		"mov.u32 s4, %4;\n\t" \
//		"add.u32 t1, s1, s2;\n\t" \
//		"shf.r.clamp.b32 t2, s3, s3, 7;\n\t" \
//		"shf.r.clamp.b32 t3, s3, s3, 18;\n\t" \
//		"xor.b32 t2, t2, t3;\n\t" \
//		"shr.u32 t3, s3, 3;\n\t" \
//		"xor.b32 t2, t2 ,t3;\n\t" \
//		"add.u32 t1, t1, t2;\n\t" \
//		"shf.r.clamp.b32 t2, s4, s4, 17;\n\t" \
//		"shf.r.clamp.b32 t3, s4, s4, 19;\n\t" \
//		"xor.b32 t2, t2, t3;\n\t" \
//		"shr.u32 t3, %4, 10;\n\t" \
//		"xor.b32 t2, t2, t3;\n\t" \
//		"add.u32 t1, t1, t2;\n\t" \
//		"mov.u32 %0, t1;\n\t" \
//		"}" \
//		: "=r"(schedule[i]) : "r"(schedule[i - 16]), "r"(schedule[i - 7]), "r"(schedule[i - 15]), "r"(schedule[i - 2]));

//#define SCHEDULE(i) \
//	asm("{\n\t" \
//		".reg .u32 t2;\n\t" \
//		".reg .u32 t3;\n\t" \
//		"shf.r.clamp.b32 t2, %3, %3, 7;\n\t" \
//		"shf.r.clamp.b32 t3, %3, %3, 18;\n\t" \
//		"xor.b32 t2, t2, t3;\n\t" \
//		"shr.u32 t3, %3, 3;\n\t" \
//		"xor.b32 t2, t2 ,t3;\n\t" \
//		"add.u32 %0, %0, t2;\n\t" \
//		"shf.r.clamp.b32 t2, %4, %4, 17;\n\t" \
//		"shf.r.clamp.b32 t3, %4, %4, 19;\n\t" \
//		"xor.b32 t2, t2, t3;\n\t" \
//		"shr.u32 t3, %4, 10;\n\t" \
//		"xor.b32 t2, t2, t3;\n\t" \
//		"add.u32 %0, %0, t2;\n\t" \
//		"add.u32 t2, %1, %2;\n\t" \
//		"add.u32 %0, %0, t2;\n\t" \
//		"}" \
//		: "=r"(schedule[i]) : "r"(schedule[i - 16]), "r"(schedule[i - 7]), "r"(schedule[i - 15]), "r"(schedule[i - 2]));

#define ROUND(a, b, c, d, e, f, g, h, i, k) \
		h = 0U + h + (ROTR32(e, 6) ^ ROTR32(e, 11) ^ ROTR32(e, 25)) + (g ^ (e & (f ^ g))) + UINT32_C(k) + schedule[i];  \
		d = 0U + d + h;  \
		h = 0U + h + (ROTR32(a, 2) ^ ROTR32(a, 13) ^ ROTR32(a, 22)) + ((a & (b | c)) | (b & c));

#define CUDA_CALL(x) {const hipError_t a = (x);if(a!=hipSuccess){printf("\nCUDA Error:%s(err_num=%d)\n",hipGetErrorString(a),a);}}
#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \ printf("Error at %s:%d\n",__FILE__,__LINE__);\ return EXIT_FAILURE;}} while(0)

//__shared__ uint k[64];

__constant__ char constantAreaPlainCharSet[36];

__shared__ char plainCharSet[384][95];

__shared__ uint state[384][8];

struct ChainComparator {
	__host__ __device__
		bool operator()(const struct Chain& lhs, const struct Chain& rhs) {
		return lhs.indexE < rhs.indexE;
	}
};

__device__ void indexToPlain(ulong index, size_t plainCharsetSize,
	size_t plainLength, char* plain)
{
	char * plainCharSetP = plainCharSet[threadIdx.x];
	for (size_t i = 0;i < plainLength;i++) {
		plain[i] = plainCharSetP[index % plainCharsetSize];
		index /= plainCharsetSize;
	}
}

__device__ inline void indexToPlain(ulong index, char* plain, const uint8_t plainLength, const char* charSet, const unsigned int charSetSize)
{
	for (int i = plainLength - 1; i >= 0; i--) {
		plain[i] = charSet[(index & 0x7f) % charSetSize];
		index >>= 7;
	}
}

/*__device__ ulong plainToIndex(const char* plain, size_t plainLength, const char* charSet, size_t charSetSize, map<char, size_t>* charIndexMap)
{
ulong index = 0;
int i;

for (i = 0;i<plainLength - 1;i++) {
index += charIndexMap->operator[](plain[i]) & 0x7f;
index <<= 7;
}
index += charIndexMap->operator[](plain[i]) & 0x7f;
return index;
}*/

__device__ inline ulong hashToIndexPaperVersion(unsigned char* hash, int pos, const uint8_t plainCharSetSize)
{
	unsigned int* hashP = (unsigned int*)hash;
	unsigned int p0 = *(hashP + 4) ^ pos;
	unsigned int p2 = *(hashP + 5) ^ (pos >> 12);
	unsigned int p4 = *(hashP + 6) ^ (pos >> 24);
	unsigned int p6 = *(hashP + 7);
	char* plainCharSetP = plainCharSet[threadIdx.x];

	unsigned __int16 b0 = plainCharSetP[p0 % plainCharSetSize] << 8 | plainCharSetP[(p0 >> 16) % plainCharSetSize];
	unsigned __int16 b1 = plainCharSetP[p2 % plainCharSetSize] << 8 | plainCharSetP[(p2 >> 16) % plainCharSetSize];
	unsigned __int16 b2 = plainCharSetP[p4 % plainCharSetSize] << 8 | plainCharSetP[(p4 >> 16) % plainCharSetSize];
	unsigned __int16 b3 = plainCharSetP[p6 % plainCharSetSize] << 8 | plainCharSetP[(p6 >> 16) % plainCharSetSize];
	/*unsigned __int16 b0 = constantAreaPlainCharSet[p0 % plainCharSetSize] << 8 | constantAreaPlainCharSet[(p0 >> 16) % plainCharSetSize];
	unsigned __int16 b1 = constantAreaPlainCharSet[p2 % plainCharSetSize] << 8 | constantAreaPlainCharSet[(p2 >> 16) % plainCharSetSize];
	unsigned __int16 b2 = constantAreaPlainCharSet[p4 % plainCharSetSize] << 8 | constantAreaPlainCharSet[(p4 >> 16) % plainCharSetSize];
	unsigned __int16 b3 = constantAreaPlainCharSet[p6 % plainCharSetSize] << 8 | constantAreaPlainCharSet[(p6 >> 16) % plainCharSetSize];*/
	ulong index = 0;
	index += b0;
	index <<= 16;
	index += b1;
	index <<= 16;
	index += b2;
	index <<= 16;
	index += b3;
	return index;
}

__device__ inline ulong hashToIndexWithoutCharSet(unsigned char* hash, int pos, const uint8_t plainCharSetSize)
{
	unsigned int* hashP = (unsigned int*)hash;
	unsigned int p0 = *(hashP + 4) ^ pos;
	unsigned int p2 = *(hashP + 5) ^ (pos >> 12);
	unsigned int p4 = *(hashP + 6) ^ (pos >> 24);
	unsigned int p6 = *(hashP + 7);

	unsigned __int16 b0 = ((p0 % plainCharSetSize) << 8) | ((p0 >> 16) % plainCharSetSize);
	unsigned __int16 b1 = ((p2 % plainCharSetSize) << 8) | ((p2 >> 16) % plainCharSetSize);
	unsigned __int16 b2 = ((p4 % plainCharSetSize) << 8) | ((p4 >> 16) % plainCharSetSize);
	unsigned __int16 b3 = ((p6 % plainCharSetSize) << 8) | ((p6 >> 16) % plainCharSetSize);

	ulong index = 0;
	index += b0;
	index <<= 16;
	index += b1;
	index <<= 16;
	index += b2;
	index <<= 16;
	index += b3;
	return index;
}


//__device__ inline void plainToHash(char* plain, const size_t length, unsigned char* res)
//{
//	unsigned int bitlen0 = 0;
//	unsigned int bitlen1 = 0;
//	//unsigned int stateP[8];
//
//	unsigned char data[64];
//
//	unsigned int l;
//
//	for (l = 0; l < length; ++l) {
//		data[l] = plain[l];
//	}
//
//	uint* stateP = state[threadIdx.x];
//
//	stateP[0] = 0x6a09e667;
//	stateP[1] = 0xbb67ae85;
//	stateP[2] = 0x3c6ef372;
//	stateP[3] = 0xa54ff53a;
//	stateP[4] = 0x510e527f;
//	stateP[5] = 0x9b05688c;
//	stateP[6] = 0x1f83d9ab;
//	stateP[7] = 0x5be0cd19;
//
//
//	// Pad whatever data is left in the buffer. 
//	data[l++] = 0x80;
//	while (l < 56)
//		data[l++] = 0x00;
//
//
//	// Append to the padding the total message's length in bits and transform. 
//	DBL_INT_ADD(bitlen0, bitlen1, length * 8);
//	data[63] = bitlen0;
//	data[62] = bitlen0 >> 8;
//	data[61] = bitlen0 >> 16;
//	data[60] = bitlen0 >> 24;
//	data[59] = bitlen1;
//	data[58] = bitlen1 >> 8;
//	data[57] = bitlen1 >> 16;
//	data[56] = bitlen1 >> 24;
//
//	unsigned int a, b, c, d, e, f, g, h, i, j, t1, t2, m[64];
//
//	for (i = 0, j = 0; i < 16; ++i, j += 4)
//		m[i] = (data[j] << 24) | (data[j + 1] << 16) | (data[j + 2] << 8) | (data[j + 3]);
//	for (; i < 64; ++i)
//		m[i] = SIG1(m[i - 2]) + m[i - 7] + SIG0(m[i - 15]) + m[i - 16];
//
//	a = stateP[0];
//	b = stateP[1];
//	c = stateP[2];
//	d = stateP[3];
//	e = stateP[4];
//	f = stateP[5];
//	g = stateP[6];
//	h = stateP[7];
//
//	for (i = 0; i < 64; ++i) {
//		t1 = h + EP1(e) + CH(e, f, g) + k[i] + m[i];
//		t2 = EP0(a) + MAJ(a, b, c);
//		h = g;
//		g = f;
//		f = e;
//		e = d + t1;
//		d = c;
//		c = b;
//		b = a;
//		a = t1 + t2;
//	}
//
//	stateP[0] += a;
//	stateP[1] += b;
//	stateP[2] += c;
//	stateP[3] += d;
//	stateP[4] += e;
//	stateP[5] += f;
//	stateP[6] += g;
//	stateP[7] += h;
//
//	// Since this implementation uses little endian byte ordering and SHA uses big endian,
//	// reverse all the bytes when copying the final state to the output hash. 
//
//	for (i = 0; i < 4; ++i) {
//		l = i << 3;
//		*(res) = (stateP[0] >> (24 - l)) & 0x000000ff;
//		*(res + 4) = (stateP[1] >> (24 - l)) & 0x000000ff;
//		*(res + 8) = (stateP[2] >> (24 - l)) & 0x000000ff;
//		*(res + 12) = (stateP[3] >> (24 - l)) & 0x000000ff;
//		*(res + 16) = (stateP[4] >> (24 - l)) & 0x000000ff;
//		*(res + 20) = (stateP[5] >> (24 - l)) & 0x000000ff;
//		*(res + 24) = (stateP[6] >> (24 - l)) & 0x000000ff;
//		*(res + 28) = (stateP[7] >> (24 - l)) & 0x000000ff;
//		++res;
//	}
//}


__device__ inline void plainToHashWithInlinePTX(char* plain, const unsigned int length, unsigned char* res) {
	unsigned int bitlen0 = 0;
	unsigned int bitlen1 = 0;
	unsigned int stateP[8];

	//unsigned int* stateP = state[threadIdx.x];

	unsigned char data[64];

	unsigned int l;

	for (l = 0; l < length; ++l) {
		data[l] = plain[l];
	}


	stateP[0] = 0x6a09e667;
	stateP[1] = 0xbb67ae85;
	stateP[2] = 0x3c6ef372;
	stateP[3] = 0xa54ff53a;
	stateP[4] = 0x510e527f;
	stateP[5] = 0x9b05688c;
	stateP[6] = 0x1f83d9ab;
	stateP[7] = 0x5be0cd19;


	data[l++] = 0x80;
	while (l < 56)
		data[l++] = 0x00;


	//// Append to the padding the total message's length in bits and transform. 
	DBL_INT_ADD(bitlen0, bitlen1, length * 8);

	data[63] = bitlen0;
	data[62] = bitlen0 >> 8;
	data[61] = bitlen0 >> 16;
	data[60] = bitlen0 >> 24;
	data[59] = bitlen1;
	data[58] = bitlen1 >> 8;
	data[57] = bitlen1 >> 16;
	data[56] = bitlen1 >> 24;

	uint32_t schedule[16];

	uint32_t a = stateP[0];
	uint32_t b = stateP[1];
	uint32_t c = stateP[2];
	uint32_t d = stateP[3];
	uint32_t e = stateP[4];
	uint32_t f = stateP[5];
	uint32_t g = stateP[6];
	uint32_t h = stateP[7];

	ROUNDa(0, a, b, c, d, e, f, g, h, 0x428A2F98)
	ROUNDa(1, h, a, b, c, d, e, f, g, 0x71374491)
	ROUNDa(2, g, h, a, b, c, d, e, f, -0x4A3F0431)
	ROUNDa(3, f, g, h, a, b, c, d, e, -0x164A245B)
	ROUNDa(4, e, f, g, h, a, b, c, d, 0x3956C25B)
	ROUNDa(5, d, e, f, g, h, a, b, c, 0x59F111F1)
	ROUNDa(6, c, d, e, f, g, h, a, b, -0x6DC07D5C)
	ROUNDa(7, b, c, d, e, f, g, h, a, -0x54E3A12B)
	ROUNDa(8, a, b, c, d, e, f, g, h, -0x27F85568)
	ROUNDa(9, h, a, b, c, d, e, f, g, 0x12835B01)
	ROUNDa(10, g, h, a, b, c, d, e, f, 0x243185BE)
	ROUNDa(11, f, g, h, a, b, c, d, e, 0x550C7DC3)
	ROUNDa(12, e, f, g, h, a, b, c, d, 0x72BE5D74)
	ROUNDa(13, d, e, f, g, h, a, b, c, -0x7F214E02)
	ROUNDa(14, c, d, e, f, g, h, a, b, -0x6423F959)
	ROUNDa(15, b, c, d, e, f, g, h, a, -0x3E640E8C)
	ROUND16(16, a, b, c, d, e, f, g, h, -0x1B64963F)
	ROUND17(17, h, a, b, c, d, e, f, g, -0x1041B87A)
	ROUND18(18, g, h, a, b, c, d, e, f, 0x0FC19DC6)
	ROUND19(19, f, g, h, a, b, c, d, e, 0x240CA1CC)
	ROUND20(20, e, f, g, h, a, b, c, d, 0x2DE92C6F)
	ROUND21(21, d, e, f, g, h, a, b, c, 0x4A7484AA)
	ROUND22(22, c, d, e, f, g, h, a, b, 0x5CB0A9DC)
	ROUND23(23, b, c, d, e, f, g, h, a, 0x76F988DA)
	ROUND24(24, a, b, c, d, e, f, g, h, -0x67C1AEAE)
	ROUND25(25, h, a, b, c, d, e, f, g, -0x57CE3993)
	ROUND26(26, g, h, a, b, c, d, e, f, -0x4FFCD838)
	ROUND27(27, f, g, h, a, b, c, d, e, -0x40A68039)
	ROUND28(28, e, f, g, h, a, b, c, d, -0x391FF40D)
	ROUND29(29, d, e, f, g, h, a, b, c, -0x2A586EB9)
	ROUND30(30, c, d, e, f, g, h, a, b, 0x06CA6351)
	ROUND31(31, b, c, d, e, f, g, h, a, 0x14292967)
	ROUND16(32, a, b, c, d, e, f, g, h, 0x27B70A85)
	ROUND17(33, h, a, b, c, d, e, f, g, 0x2E1B2138)
	ROUND18(34, g, h, a, b, c, d, e, f, 0x4D2C6DFC)
	ROUND19(35, f, g, h, a, b, c, d, e, 0x53380D13)
	ROUND20(36, e, f, g, h, a, b, c, d, 0x650A7354)
	ROUND21(37, d, e, f, g, h, a, b, c, 0x766A0ABB)
	ROUND22(38, c, d, e, f, g, h, a, b, -0x7E3D36D2)
	ROUND23(39, b, c, d, e, f, g, h, a, -0x6D8DD37B)
	ROUND24(40, a, b, c, d, e, f, g, h, -0x5D40175F)
	ROUND25(41, h, a, b, c, d, e, f, g, -0x57E599B5)
	ROUND26(42, g, h, a, b, c, d, e, f, -0x3DB47490)
	ROUND27(43, f, g, h, a, b, c, d, e, -0x3893AE5D)
	ROUND28(44, e, f, g, h, a, b, c, d, -0x2E6D17E7)
	ROUND29(45, d, e, f, g, h, a, b, c, -0x2966F9DC)
	ROUND30(46, c, d, e, f, g, h, a, b, -0x0BF1CA7B)
	ROUND31(47, b, c, d, e, f, g, h, a, 0x106AA070)
	ROUND16(48, a, b, c, d, e, f, g, h, 0x19A4C116)
	ROUND17(49, h, a, b, c, d, e, f, g, 0x1E376C08)
	ROUND18(50, g, h, a, b, c, d, e, f, 0x2748774C)
	ROUND19(51, f, g, h, a, b, c, d, e, 0x34B0BCB5)
	ROUND20(52, e, f, g, h, a, b, c, d, 0x391C0CB3)
	ROUND21(53, d, e, f, g, h, a, b, c, 0x4ED8AA4A)
	ROUND22(54, c, d, e, f, g, h, a, b, 0x5B9CCA4F)
	ROUND23(55, b, c, d, e, f, g, h, a, 0x682E6FF3)
	ROUND24(56, a, b, c, d, e, f, g, h, 0x748F82EE)
	ROUND25(57, h, a, b, c, d, e, f, g, 0x78A5636F)
	ROUND26(58, g, h, a, b, c, d, e, f, -0x7B3787EC)
	ROUND27(59, f, g, h, a, b, c, d, e, -0x7338FDF8)
	ROUND28(60, e, f, g, h, a, b, c, d, -0x6F410006)
	ROUND29(61, d, e, f, g, h, a, b, c, -0x5BAF9315)
	ROUND30(62, c, d, e, f, g, h, a, b, -0x41065C09)
	ROUND31(63, b, c, d, e, f, g, h, a, -0x398E870E)

	stateP[0] += a;
	stateP[1] += b;
	stateP[2] += c;
	stateP[3] += d;
	stateP[4] += e;
	stateP[5] += f;
	stateP[6] += g;
	stateP[7] += h;

	// Since this implementation uses little endian byte ordering and SHA uses big endian,
	// reverse all the bytes when copying the final state to the output hash. 

	for (int i = 0; i < 4; ++i) {
		l = i << 3;
		*(res) = (stateP[0] >> (24 - l)) & 0x000000ff;
		*(res + 4) = (stateP[1] >> (24 - l)) & 0x000000ff;
		*(res + 8) = (stateP[2] >> (24 - l)) & 0x000000ff;
		*(res + 12) = (stateP[3] >> (24 - l)) & 0x000000ff;
		*(res + 16) = (stateP[4] >> (24 - l)) & 0x000000ff;
		*(res + 20) = (stateP[5] >> (24 - l)) & 0x000000ff;
		*(res + 24) = (stateP[6] >> (24 - l)) & 0x000000ff;
		*(res + 28) = (stateP[7] >> (24 - l)) & 0x000000ff;
		++res;
	}

	//unsigned int* resP = (unsigned int*)res;
	//unsigned char* stateCP = (unsigned char*)stateP;

	//*(resP) = (((unsigned int)*(stateCP)<<0)| ((unsigned int)*(stateCP+1)<<8)| ((unsigned int)*(stateCP+2)<<16)| ((unsigned int)*(stateCP+3)<<24));
	//*(resP+1) = ((unsigned int)(*(stateCP+4) << 0) | ((unsigned int)*(stateCP + 5) << 8) | ((unsigned int)*(stateCP + 6) << 16) | ((unsigned int)*(stateCP + 7) << 24));
	//*(resP+2) = (((unsigned int)*(stateCP+8) << 0) | ((unsigned int)*(stateCP + 9) << 8) | ((unsigned int)*(stateCP + 10) << 16) | ((unsigned int)*(stateCP + 11) << 24));
	//*(resP+3) = (((unsigned int)*(stateCP+12) << 0) | ((unsigned int)*(stateCP + 13) << 8) | ((unsigned int)*(stateCP + 14) << 16) | ((unsigned int)*(stateCP + 15) << 24));
	//*(resP+4) = (((unsigned int)*(stateCP+16) << 0) | ((unsigned int)*(stateCP + 17) << 8) | ((unsigned int)*(stateCP + 18) << 16) | ((unsigned int)*(stateCP + 19) << 24));
	//*(resP+5) = (((unsigned int)*(stateCP+20) << 0) | ((unsigned int)*(stateCP + 21) << 8) | ((unsigned int)*(stateCP + 22) << 16) | ((unsigned int)*(stateCP + 23) << 24));
	//*(resP+6) = (((unsigned int)*(stateCP+24) << 0) | ((unsigned int)*(stateCP + 25) << 8) | ((unsigned int)*(stateCP + 26) << 16) | ((unsigned int)*(stateCP + 27) << 24));
	//*(resP+7) = (((unsigned int)*(stateCP+28) << 0) | ((unsigned int)*(stateCP + 29) << 8) | ((unsigned int)*(stateCP + 30) << 16) | ((unsigned int)*(stateCP + 31) << 24));

	///**((unsigned int*)res) = ((*((unsigned char*)stateP) << 0) | (*((unsigned char*)stateP + 1) << 8) | (*((unsigned char*)stateP + 2) << 16) | (*((unsigned char*)stateP + 3) << 24));
	//*((unsigned int*)res + 1) = ((*((unsigned char*)stateP + 4) << 0) | (*((unsigned char*)stateP + 5) << 8) | (*((unsigned char*)stateP + 6) << 16) | (*((unsigned char*)stateP + 7) << 24));
	//*((unsigned int*)res + 2) = ((*((unsigned char*)stateP + 8) << 0) | (*((unsigned char*)stateP + 9) << 8) | (*((unsigned char*)stateP + 10) << 16) | (*((unsigned char*)stateP + 11) << 24));
	//*((unsigned int*)res + 3) = ((*((unsigned char*)stateP + 12) << 0) | (*((unsigned char*)stateP + 13) << 8) | (*((unsigned char*)stateP + 14) << 16) | (*((unsigned char*)stateP + 15) << 24));
	//*((unsigned int*)res + 4) = ((*((unsigned char*)stateP + 16) << 0) | (*((unsigned char*)stateP + 17) << 8) | (*((unsigned char*)stateP + 18) << 16) | (*((unsigned char*)stateP + 19) << 24));
	//*((unsigned int*)res + 5) = ((*((unsigned char*)stateP + 20) << 0) | (*((unsigned char*)stateP + 21) << 8) | (*((unsigned char*)stateP + 22) << 16) | (*((unsigned char*)stateP + 23) << 24));
	//*((unsigned int*)res + 6) = ((*((unsigned char*)stateP + 24) << 0) | (*((unsigned char*)stateP + 25) << 8) | (*((unsigned char*)stateP + 26) << 16) | (*((unsigned char*)stateP + 27) << 24));
	//*((unsigned int*)res + 7) = ((*((unsigned char*)stateP + 28) << 0) | (*((unsigned char*)stateP + 29) << 8) | (*((unsigned char*)stateP + 30) << 16) | (*((unsigned char*)stateP + 31) << 24));*/
}

__device__ inline void plainToHashWithInlinePTX(ulong index, const uint8_t length, unsigned char* res, const uint8_t charSetSize) {
	unsigned int bitlen0 = 0;
	unsigned int bitlen1 = 0;
	unsigned int stateP[8];

	unsigned char data[64];

	unsigned int l;

	for (l = length - 1; l >= 1; l--) {
		data[l] = (index & 0x7f) % charSetSize;
		index >>= 7;
	}
	data[0] = (index & 0x7f) % charSetSize;
	l = length;

	stateP[0] = 0x6a09e667;
	stateP[1] = 0xbb67ae85;
	stateP[2] = 0x3c6ef372;
	stateP[3] = 0xa54ff53a;
	stateP[4] = 0x510e527f;
	stateP[5] = 0x9b05688c;
	stateP[6] = 0x1f83d9ab;
	stateP[7] = 0x5be0cd19;


	data[l++] = 0x80;
	while (l < 56)
		data[l++] = 0x00;


	//// Append to the padding the total message's length in bits and transform. 
	DBL_INT_ADD(bitlen0, bitlen1, length * 8);

	data[63] = bitlen0;
	data[62] = bitlen0 >> 8;
	data[61] = bitlen0 >> 16;
	data[60] = bitlen0 >> 24;
	data[59] = bitlen1;
	data[58] = bitlen1 >> 8;
	data[57] = bitlen1 >> 16;
	data[56] = bitlen1 >> 24;

	uint32_t schedule[16];

	uint32_t a = stateP[0];
	uint32_t b = stateP[1];
	uint32_t c = stateP[2];
	uint32_t d = stateP[3];
	uint32_t e = stateP[4];
	uint32_t f = stateP[5];
	uint32_t g = stateP[6];
	uint32_t h = stateP[7];

	ROUNDa(0, a, b, c, d, e, f, g, h, 0x428A2F98)
	ROUNDa(1, h, a, b, c, d, e, f, g, 0x71374491)
	ROUNDa(2, g, h, a, b, c, d, e, f, -0x4A3F0431)
	ROUNDa(3, f, g, h, a, b, c, d, e, -0x164A245B)
	ROUNDa(4, e, f, g, h, a, b, c, d, 0x3956C25B)
	ROUNDa(5, d, e, f, g, h, a, b, c, 0x59F111F1)
	ROUNDa(6, c, d, e, f, g, h, a, b, -0x6DC07D5C)
	ROUNDa(7, b, c, d, e, f, g, h, a, -0x54E3A12B)
	ROUNDa(8, a, b, c, d, e, f, g, h, -0x27F85568)
	ROUNDa(9, h, a, b, c, d, e, f, g, 0x12835B01)
	ROUNDa(10, g, h, a, b, c, d, e, f, 0x243185BE)
	ROUNDa(11, f, g, h, a, b, c, d, e, 0x550C7DC3)
	ROUNDa(12, e, f, g, h, a, b, c, d, 0x72BE5D74)
	ROUNDa(13, d, e, f, g, h, a, b, c, -0x7F214E02)
	ROUNDa(14, c, d, e, f, g, h, a, b, -0x6423F959)
	ROUNDa(15, b, c, d, e, f, g, h, a, -0x3E640E8C)
	ROUND16(16, a, b, c, d, e, f, g, h, -0x1B64963F)
	ROUND17(17, h, a, b, c, d, e, f, g, -0x1041B87A)
	ROUND18(18, g, h, a, b, c, d, e, f, 0x0FC19DC6)
	ROUND19(19, f, g, h, a, b, c, d, e, 0x240CA1CC)
	ROUND20(20, e, f, g, h, a, b, c, d, 0x2DE92C6F)
	ROUND21(21, d, e, f, g, h, a, b, c, 0x4A7484AA)
	ROUND22(22, c, d, e, f, g, h, a, b, 0x5CB0A9DC)
	ROUND23(23, b, c, d, e, f, g, h, a, 0x76F988DA)
	ROUND24(24, a, b, c, d, e, f, g, h, -0x67C1AEAE)
	ROUND25(25, h, a, b, c, d, e, f, g, -0x57CE3993)
	ROUND26(26, g, h, a, b, c, d, e, f, -0x4FFCD838)
	ROUND27(27, f, g, h, a, b, c, d, e, -0x40A68039)
	ROUND28(28, e, f, g, h, a, b, c, d, -0x391FF40D)
	ROUND29(29, d, e, f, g, h, a, b, c, -0x2A586EB9)
	ROUND30(30, c, d, e, f, g, h, a, b, 0x06CA6351)
	ROUND31(31, b, c, d, e, f, g, h, a, 0x14292967)
	ROUND16(32, a, b, c, d, e, f, g, h, 0x27B70A85)
	ROUND17(33, h, a, b, c, d, e, f, g, 0x2E1B2138)
	ROUND18(34, g, h, a, b, c, d, e, f, 0x4D2C6DFC)
	ROUND19(35, f, g, h, a, b, c, d, e, 0x53380D13)
	ROUND20(36, e, f, g, h, a, b, c, d, 0x650A7354)
	ROUND21(37, d, e, f, g, h, a, b, c, 0x766A0ABB)
	ROUND22(38, c, d, e, f, g, h, a, b, -0x7E3D36D2)
	ROUND23(39, b, c, d, e, f, g, h, a, -0x6D8DD37B)
	ROUND24(40, a, b, c, d, e, f, g, h, -0x5D40175F)
	ROUND25(41, h, a, b, c, d, e, f, g, -0x57E599B5)
	ROUND26(42, g, h, a, b, c, d, e, f, -0x3DB47490)
	ROUND27(43, f, g, h, a, b, c, d, e, -0x3893AE5D)
	ROUND28(44, e, f, g, h, a, b, c, d, -0x2E6D17E7)
	ROUND29(45, d, e, f, g, h, a, b, c, -0x2966F9DC)
	ROUND30(46, c, d, e, f, g, h, a, b, -0x0BF1CA7B)
	ROUND31(47, b, c, d, e, f, g, h, a, 0x106AA070)
	ROUND16(48, a, b, c, d, e, f, g, h, 0x19A4C116)
	ROUND17(49, h, a, b, c, d, e, f, g, 0x1E376C08)
	ROUND18(50, g, h, a, b, c, d, e, f, 0x2748774C)
	ROUND19(51, f, g, h, a, b, c, d, e, 0x34B0BCB5)
	ROUND20(52, e, f, g, h, a, b, c, d, 0x391C0CB3)
	ROUND21(53, d, e, f, g, h, a, b, c, 0x4ED8AA4A)
	ROUND22(54, c, d, e, f, g, h, a, b, 0x5B9CCA4F)
	ROUND23(55, b, c, d, e, f, g, h, a, 0x682E6FF3)
	ROUND24(56, a, b, c, d, e, f, g, h, 0x748F82EE)
	ROUND25(57, h, a, b, c, d, e, f, g, 0x78A5636F)
	ROUND26(58, g, h, a, b, c, d, e, f, -0x7B3787EC)
	ROUND27(59, f, g, h, a, b, c, d, e, -0x7338FDF8)
	ROUND28(60, e, f, g, h, a, b, c, d, -0x6F410006)
	ROUND29(61, d, e, f, g, h, a, b, c, -0x5BAF9315)
	ROUND30(62, c, d, e, f, g, h, a, b, -0x41065C09)
	ROUND31(63, b, c, d, e, f, g, h, a, -0x398E870E)

	stateP[0] += a;
	stateP[1] += b;
	stateP[2] += c;
	stateP[3] += d;
	stateP[4] += e;
	stateP[5] += f;
	stateP[6] += g;
	stateP[7] += h;

	// Since this implementation uses little endian byte ordering and SHA uses big endian,
	// reverse all the bytes when copying the final state to the output hash. 

	for (int i = 0; i < 4; ++i) {
		l = i << 3;
		*(res) = (stateP[0] >> (24 - l)) & 0x000000ff;
		*(res + 4) = (stateP[1] >> (24 - l)) & 0x000000ff;
		*(res + 8) = (stateP[2] >> (24 - l)) & 0x000000ff;
		*(res + 12) = (stateP[3] >> (24 - l)) & 0x000000ff;
		*(res + 16) = (stateP[4] >> (24 - l)) & 0x000000ff;
		*(res + 20) = (stateP[5] >> (24 - l)) & 0x000000ff;
		*(res + 24) = (stateP[6] >> (24 - l)) & 0x000000ff;
		*(res + 28) = (stateP[7] >> (24 - l)) & 0x000000ff;
		++res;
	}
}

__device__ inline void plainToHash(char* plain, const uint8_t length, unsigned char* res)
{
	unsigned int bitlen0 = 0;
	unsigned int bitlen1 = 0;
	//unsigned int stateP[8];

	unsigned char data[64];

	unsigned int l;

	for (l = 0; l < length; ++l) {
		data[l] = plain[l];
	}

	uint* stateP = state[threadIdx.x];

	stateP[0] = 0x6a09e667;
	stateP[1] = 0xbb67ae85;
	stateP[2] = 0x3c6ef372;
	stateP[3] = 0xa54ff53a;
	stateP[4] = 0x510e527f;
	stateP[5] = 0x9b05688c;
	stateP[6] = 0x1f83d9ab;
	stateP[7] = 0x5be0cd19;


	// Pad whatever data is left in the buffer. 
	data[l++] = 0x80;
	while (l < 56)
		data[l++] = 0x00;


	// Append to the padding the total message's length in bits and transform. 
	DBL_INT_ADD(bitlen0, bitlen1, length * 8);
	data[63] = bitlen0;
	data[62] = bitlen0 >> 8;
	data[61] = bitlen0 >> 16;
	data[60] = bitlen0 >> 24;
	data[59] = bitlen1;
	data[58] = bitlen1 >> 8;
	data[57] = bitlen1 >> 16;
	data[56] = bitlen1 >> 24;


	uint32_t schedule[64];
	LOADSCHEDULE(0)
	LOADSCHEDULE(1)
	LOADSCHEDULE(2)
	LOADSCHEDULE(3)
	LOADSCHEDULE(4)
	LOADSCHEDULE(5)
	LOADSCHEDULE(6)
	LOADSCHEDULE(7)
	LOADSCHEDULE(8)
	LOADSCHEDULE(9)
	LOADSCHEDULE(10)
	LOADSCHEDULE(11)
	LOADSCHEDULE(12)
	LOADSCHEDULE(13)
	LOADSCHEDULE(14)
	LOADSCHEDULE(15)
	SCHEDULE(16)
	SCHEDULE(17)
	SCHEDULE(18)
	SCHEDULE(19)
	SCHEDULE(20)
	SCHEDULE(21)
	SCHEDULE(22)
	SCHEDULE(23)
	SCHEDULE(24)
	SCHEDULE(25)
	SCHEDULE(26)
	SCHEDULE(27)
	SCHEDULE(28)
	SCHEDULE(29)
	SCHEDULE(30)
	SCHEDULE(31)
	SCHEDULE(32)
	SCHEDULE(33)
	SCHEDULE(34)
	SCHEDULE(35)
	SCHEDULE(36)
	SCHEDULE(37)
	SCHEDULE(38)
	SCHEDULE(39)
	SCHEDULE(40)
	SCHEDULE(41)
	SCHEDULE(42)
	SCHEDULE(43)
	SCHEDULE(44)
	SCHEDULE(45)
	SCHEDULE(46)
	SCHEDULE(47)
	SCHEDULE(48)
	SCHEDULE(49)
	SCHEDULE(50)
	SCHEDULE(51)
	SCHEDULE(52)
	SCHEDULE(53)
	SCHEDULE(54)
	SCHEDULE(55)
	SCHEDULE(56)
	SCHEDULE(57)
	SCHEDULE(58)
	SCHEDULE(59)
	SCHEDULE(60)
	SCHEDULE(61)
	SCHEDULE(62)
	SCHEDULE(63)

	uint32_t a = stateP[0];
	uint32_t b = stateP[1];
	uint32_t c = stateP[2];
	uint32_t d = stateP[3];
	uint32_t e = stateP[4];
	uint32_t f = stateP[5];
	uint32_t g = stateP[6];
	uint32_t h = stateP[7];
	ROUND(a, b, c, d, e, f, g, h, 0, 0x428A2F98)
	ROUND(h, a, b, c, d, e, f, g, 1, 0x71374491)
	ROUND(g, h, a, b, c, d, e, f, 2, 0xB5C0FBCF)
	ROUND(f, g, h, a, b, c, d, e, 3, 0xE9B5DBA5)
	ROUND(e, f, g, h, a, b, c, d, 4, 0x3956C25B)
	ROUND(d, e, f, g, h, a, b, c, 5, 0x59F111F1)
	ROUND(c, d, e, f, g, h, a, b, 6, 0x923F82A4)
	ROUND(b, c, d, e, f, g, h, a, 7, 0xAB1C5ED5)
	ROUND(a, b, c, d, e, f, g, h, 8, 0xD807AA98)
	ROUND(h, a, b, c, d, e, f, g, 9, 0x12835B01)
	ROUND(g, h, a, b, c, d, e, f, 10, 0x243185BE)
	ROUND(f, g, h, a, b, c, d, e, 11, 0x550C7DC3)
	ROUND(e, f, g, h, a, b, c, d, 12, 0x72BE5D74)
	ROUND(d, e, f, g, h, a, b, c, 13, 0x80DEB1FE)
	ROUND(c, d, e, f, g, h, a, b, 14, 0x9BDC06A7)
	ROUND(b, c, d, e, f, g, h, a, 15, 0xC19BF174)
	ROUND(a, b, c, d, e, f, g, h, 16, 0xE49B69C1)
	ROUND(h, a, b, c, d, e, f, g, 17, 0xEFBE4786)
	ROUND(g, h, a, b, c, d, e, f, 18, 0x0FC19DC6)
	ROUND(f, g, h, a, b, c, d, e, 19, 0x240CA1CC)
	ROUND(e, f, g, h, a, b, c, d, 20, 0x2DE92C6F)
	ROUND(d, e, f, g, h, a, b, c, 21, 0x4A7484AA)
	ROUND(c, d, e, f, g, h, a, b, 22, 0x5CB0A9DC)
	ROUND(b, c, d, e, f, g, h, a, 23, 0x76F988DA)
	ROUND(a, b, c, d, e, f, g, h, 24, 0x983E5152)
	ROUND(h, a, b, c, d, e, f, g, 25, 0xA831C66D)
	ROUND(g, h, a, b, c, d, e, f, 26, 0xB00327C8)
	ROUND(f, g, h, a, b, c, d, e, 27, 0xBF597FC7)
	ROUND(e, f, g, h, a, b, c, d, 28, 0xC6E00BF3)
	ROUND(d, e, f, g, h, a, b, c, 29, 0xD5A79147)
	ROUND(c, d, e, f, g, h, a, b, 30, 0x06CA6351)
	ROUND(b, c, d, e, f, g, h, a, 31, 0x14292967)
	ROUND(a, b, c, d, e, f, g, h, 32, 0x27B70A85)
	ROUND(h, a, b, c, d, e, f, g, 33, 0x2E1B2138)
	ROUND(g, h, a, b, c, d, e, f, 34, 0x4D2C6DFC)
	ROUND(f, g, h, a, b, c, d, e, 35, 0x53380D13)
	ROUND(e, f, g, h, a, b, c, d, 36, 0x650A7354)
	ROUND(d, e, f, g, h, a, b, c, 37, 0x766A0ABB)
	ROUND(c, d, e, f, g, h, a, b, 38, 0x81C2C92E)
	ROUND(b, c, d, e, f, g, h, a, 39, 0x92722C85)
	ROUND(a, b, c, d, e, f, g, h, 40, 0xA2BFE8A1)
	ROUND(h, a, b, c, d, e, f, g, 41, 0xA81A664B)
	ROUND(g, h, a, b, c, d, e, f, 42, 0xC24B8B70)
	ROUND(f, g, h, a, b, c, d, e, 43, 0xC76C51A3)
	ROUND(e, f, g, h, a, b, c, d, 44, 0xD192E819)
	ROUND(d, e, f, g, h, a, b, c, 45, 0xD6990624)
	ROUND(c, d, e, f, g, h, a, b, 46, 0xF40E3585)
	ROUND(b, c, d, e, f, g, h, a, 47, 0x106AA070)
	ROUND(a, b, c, d, e, f, g, h, 48, 0x19A4C116)
	ROUND(h, a, b, c, d, e, f, g, 49, 0x1E376C08)
	ROUND(g, h, a, b, c, d, e, f, 50, 0x2748774C)
	ROUND(f, g, h, a, b, c, d, e, 51, 0x34B0BCB5)
	ROUND(e, f, g, h, a, b, c, d, 52, 0x391C0CB3)
	ROUND(d, e, f, g, h, a, b, c, 53, 0x4ED8AA4A)
	ROUND(c, d, e, f, g, h, a, b, 54, 0x5B9CCA4F)
	ROUND(b, c, d, e, f, g, h, a, 55, 0x682E6FF3)
	ROUND(a, b, c, d, e, f, g, h, 56, 0x748F82EE)
	ROUND(h, a, b, c, d, e, f, g, 57, 0x78A5636F)
	ROUND(g, h, a, b, c, d, e, f, 58, 0x84C87814)
	ROUND(f, g, h, a, b, c, d, e, 59, 0x8CC70208)
	ROUND(e, f, g, h, a, b, c, d, 60, 0x90BEFFFA)
	ROUND(d, e, f, g, h, a, b, c, 61, 0xA4506CEB)
	ROUND(c, d, e, f, g, h, a, b, 62, 0xBEF9A3F7)
	ROUND(b, c, d, e, f, g, h, a, 63, 0xC67178F2)
	stateP[0] += a;
	stateP[1] += b;
	stateP[2] += c;
	stateP[3] += d;
	stateP[4] += e;
	stateP[5] += f;
	stateP[6] += g;
	stateP[7] += h;

	// Since this implementation uses little endian byte ordering and SHA uses big endian,
	// reverse all the bytes when copying the final state to the output hash. 

	for (int i = 0; i < 4; ++i) {
		l = i << 3;
		*(res) = (stateP[0] >> (24 - l)) & 0x000000ff;
		*(res + 4) = (stateP[1] >> (24 - l)) & 0x000000ff;
		*(res + 8) = (stateP[2] >> (24 - l)) & 0x000000ff;
		*(res + 12) = (stateP[3] >> (24 - l)) & 0x000000ff;
		*(res + 16) = (stateP[4] >> (24 - l)) & 0x000000ff;
		*(res + 20) = (stateP[5] >> (24 - l)) & 0x000000ff;
		*(res + 24) = (stateP[6] >> (24 - l)) & 0x000000ff;
		*(res + 28) = (stateP[7] >> (24 - l)) & 0x000000ff;
		++res;
	}
}

__device__ inline void initSHA256ConstantAndCharSet(const unsigned int charSetSize)
{
	char* plainCharSetP = plainCharSet[threadIdx.x];
	//for (i = 0;i < charSetSize;i++) {
	//	plainCharSetP[i] = srcCharSet[i];
	//}
	for (int i = 0;i < charSetSize;i++) {
		plainCharSetP[i] = constantAreaPlainCharSet[i];
	}
}

__device__ inline ulong hashToIndex(unsigned char* hash, int pos)
{
	ulong* hashP = (ulong*)hash;
	return (ulong)(((*(hashP) ^ *(hashP + 1) ^ *(hashP + 2) ^ *(hashP + 3)) + pos));
}

__device__ inline ulong reductFinalIndex(ulong index, uint8_t plainLength, uint8_t plainCharSize)
{
	ulong res = 0;
	uint8_t plainIndex[9];
	for (int l = plainLength - 1; l >= 0; l--) {
		plainIndex[l] = ((uint8_t)(index & 0x7f)) % plainCharSize;
		index >>= 7;
	}
	int j;
	for (j = 0; j < plainLength - 1; j++) {
		res += plainIndex[j];
		res <<= 7;
	}
	res += plainIndex[j];
	return res;
}

__global__ void generateChainPaperVersion(struct Chain* chains, const uint8_t plainCharSetSize,
	const uint8_t plainLength, const unsigned int chainLength)
{
	//initSHA256ConstantAndCharSet(plainCharSetSize);

	unsigned char hash[32];

	uint offset = (blockIdx.x * blockDim.x) + threadIdx.x;

	struct Chain* chain;

	chain = chains + offset;

	ulong indexE = chain->indexS;

	for (int i = 0;i < chainLength;i++) {
		//plainToHashWithInlinePTX((char *)&indexE, INDEX_SIZE_IN_BYTES, hash);
		plainToHashWithInlinePTX(indexE, plainLength, hash, plainCharSetSize);
		//indexE = hashToIndexWithoutCharSet(hash, i, plainCharSetSize);
		indexE = hashToIndex(hash, i);
	}
	chain->indexE = reductFinalIndex(indexE,plainLength,plainCharSetSize);
	//}
}

int main()
{
	const uint CHAINS_SIZE = 7680000;
	int plainLength = 6;
	int chainLength = 100000;

	int plainCharSetSize = 95;

	//hipSetDeviceFlags(hipDeviceMapHost);
	struct Chain* devicePointer;
	struct Chain* hostPointer;
	char* hostCharSet;
	char* deviceCharSet;
	//CUDA_CALL(hipHostAlloc(&hostPointer, CHAINS_SIZE * sizeof(struct Chain), hipHostMallocDefault | hipHostMallocMapped));
	//CUDA_CALL(hipHostAlloc(&hostCharSet, 36 * sizeof(char), hipHostMallocDefault | hipHostMallocMapped));
	CUDA_CALL(hipHostAlloc(&hostPointer, CHAINS_SIZE * sizeof(struct Chain), hipHostMallocDefault));
	CUDA_CALL(hipHostAlloc(&hostCharSet, plainCharSetSize * sizeof(char), hipHostMallocDefault));

	getCharSet(hostCharSet, "../charsets/ascii-32-95.txt", plainCharSetSize);

	generateInitialIndex(hostPointer, CHAINS_SIZE);

	//printf("%llu", hostPointer[0].indexS);

	CUDA_CALL(hipMalloc(&devicePointer, CHAINS_SIZE * sizeof(struct Chain)));
	CUDA_CALL(hipMalloc(&deviceCharSet, plainCharSetSize * sizeof(char)));

	CUDA_CALL(hipMemcpy(deviceCharSet, hostCharSet, plainCharSetSize * sizeof(char), hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(devicePointer, hostPointer, CHAINS_SIZE * sizeof(struct Chain), hipMemcpyHostToDevice));

	CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(constantAreaPlainCharSet), hostCharSet, sizeof(char) * plainCharSetSize));

	/*hiprandGenerator_t randGeneratorDevice;
	const ulong seed = 987654321;
	const hiprandRngType_t generatorType = HIPRAND_RNG_PSEUDO_DEFAULT;

	hiprandCreateGenerator(&randGeneratorDevice, generatorType);
	hiprandSetPseudoRandomGeneratorSeed(randGeneratorDevice, seed);
	hiprandGenerateLongLong(randGeneratorDevice, (ulong *)devicePointer, CHAINS_SIZE * 2);*/

	int threadPerBlock = 384;
	uint blockNum = CHAINS_SIZE / threadPerBlock;

	hipEvent_t startEvent;
	hipEvent_t endEvent;
	float cudaElapsedTime = 0.0f;
	hipEventCreate(&startEvent);
	hipEventCreate(&endEvent);
	hipEventRecord(startEvent, 0);

	generateChainPaperVersion << <blockNum, threadPerBlock >> > (devicePointer, plainCharSetSize, plainLength, chainLength);

	hipEventRecord(endEvent, 0);
	hipEventSynchronize(endEvent);
	hipEventElapsedTime(&cudaElapsedTime, startEvent, endEvent);

	thrust::device_ptr<struct Chain> thrustChainP(devicePointer);
	thrust::sort(thrustChainP, thrustChainP + CHAINS_SIZE, ChainComparator());

	CUDA_CALL(hipMemcpy(hostPointer, devicePointer, CHAINS_SIZE * sizeof(struct Chain), hipMemcpyDeviceToHost));

	writeToFile("../t5.rt", hostPointer, sizeof(struct Chain), CHAINS_SIZE);


	hipHostFree(hostPointer);
	hipHostFree(hostCharSet);
	hipFree(deviceCharSet);
	hipFree(devicePointer);
	//hipEventDestroy(startEvent);
	//hipEventDestroy(endEvent);

	hipDeviceReset();

	printf("%.3lf MH/S", (CHAINS_SIZE * (ulong)chainLength) / (cudaElapsedTime * 1000.0));

	getchar();



	return 0;
}
